#include "hip/hip_runtime.h"
#define UTILS_NO_DEVICE_TABLE

// Functions work differently when executed from this kernel
// This emulates the old device.iteration_type == TYPE_LIGHT checks.
#define SHADING_KERNEL
#define OPTIX_KERNEL

#include "utils.h"

extern "C" static __constant__ DeviceConstantMemory device;

#include "bsdf.cuh"
#include "directives.cuh"
#include "geometry_utils.cuh"
#include "ior_stack.cuh"
#include "math.cuh"
#include "memory.cuh"
#include "shading_kernel.cuh"
#include "toy_utils.cuh"
#include "utils.cuh"

extern "C" __global__ void __raygen__optix() {
  const int task_count  = device.ptrs.task_counts[THREAD_ID * TASK_ADDRESS_COUNT_STRIDE + TASK_ADDRESS_OFFSET_GEOMETRY];
  const int task_offset = device.ptrs.task_offsets[THREAD_ID * TASK_ADDRESS_OFFSET_STRIDE + TASK_ADDRESS_OFFSET_GEOMETRY];
  int trace_count       = device.ptrs.trace_counts[THREAD_ID];

  for (int i = 0; i < task_count; i++) {
    const ShadingTask task = load_shading_task(device.ptrs.trace_tasks + get_task_address(task_offset + i));
    const int pixel        = task.index.y * device.width + task.index.x;

    GBufferData data;
    if (task.hit_id == HIT_TYPE_TOY) {
      data = toy_generate_g_buffer(task, pixel);
    }
    else {
      data = geometry_generate_g_buffer(task, pixel);
    }

    write_normal_buffer(data.normal, pixel);

    if (!material_is_mirror(data.roughness, data.metallic))
      write_albedo_buffer(opaque_color(data.albedo), pixel);

    const bool include_emission = state_peek(pixel, STATE_FLAG_BOUNCE_LIGHTING);

    const RGBF record = load_RGBF(device.ptrs.records + pixel);

    BSDFSampleInfo bounce_info;
    vec3 bounce_ray = bsdf_sample(data, task.index, bounce_info);

    RGBF bounce_record = mul_color(record, bounce_info.weight);

    bool use_light_rays        = false;
    bool allow_bounce_lighting = false;
    if (bounce_info.is_transparent_pass) {
      const IORStackMethod ior_stack_method = (data.flags & G_BUFFER_REFRACTION_IS_INSIDE) ? IOR_STACK_METHOD_PULL : IOR_STACK_METHOD_PUSH;
      ior_stack_interact(data.ior_out, pixel, ior_stack_method);

      const float refraction_scale = (data.ior_in > data.ior_out) ? data.ior_in / data.ior_out : data.ior_out / data.ior_in;
      use_light_rays |= data.roughness * (refraction_scale - 1.0f) > 0.05f;
      allow_bounce_lighting = !use_light_rays;
    }
    else {
      use_light_rays |= ((data.metallic < 1.0f && data.albedo.a > 0.0f) || data.roughness > 0.05f);
      allow_bounce_lighting |= bounce_info.is_microfacet_based && data.roughness <= 0.05f;
    }

    if (include_emission) {
      const RGBF emission = mul_color(data.emission, record);

      write_beauty_buffer(emission, pixel);
    }

    RGBF accumulated_light = get_color(0.0f, 0.0f, 0.0f);

    if (use_light_rays) {
      if (device.ris_settings.num_light_rays) {
        for (int j = 0; j < device.ris_settings.num_light_rays; j++) {
          accumulated_light = add_color(accumulated_light, optix_compute_light_ray_geometry(data, task.index, j));
        }

        accumulated_light = scale_color(accumulated_light, 1.0f / device.ris_settings.num_light_rays);
      }

      accumulated_light = add_color(accumulated_light, optix_compute_light_ray_sun(data, task.index));
      accumulated_light = add_color(accumulated_light, optix_compute_light_ray_toy(data, task.index));
    }

    accumulated_light = mul_color(accumulated_light, record);

    write_beauty_buffer(accumulated_light, pixel);

    const float shift = (bounce_info.is_transparent_pass) ? -eps : eps;
    data.position     = add_vector(data.position, scale_vector(data.V, shift * get_length(data.position)));

    TraceTask bounce_task;
    bounce_task.origin = data.position;
    bounce_task.ray    = bounce_ray;
    bounce_task.index  = task.index;

    // This must be done after the trace rays due to some optimization in the compiler.
    // The compiler reloads these values at some point for some reason and if we overwrite
    // the values we will get garbage. I am not sure if this is a compiler bug or some undefined
    // behaviour on my side.
    if (validate_trace_task(bounce_task, bounce_record)) {
      store_trace_task(device.ptrs.trace_tasks + get_task_address(trace_count++), bounce_task);
      store_RGBF(device.ptrs.records + pixel, bounce_record);

      if (!allow_bounce_lighting) {
        state_release(pixel, STATE_FLAG_BOUNCE_LIGHTING);
        state_release(pixel, STATE_FLAG_DELTA_PATH);
      }
    }
  }

  device.ptrs.trace_counts[THREAD_ID] = trace_count;
}
