#include "hip/hip_runtime.h"
#include "brdf.cuh"
#include "math.cuh"
#include "memory.cuh"
#include "random.cuh"
#include "utils.cuh"
#include "volume.cuh"

__global__ void volume_process_events() {
  const int task_count = device.trace_count[THREAD_ID];

  for (int i = 0; i < task_count; i++) {
    const int offset    = get_task_address(i);
    TraceTask task      = load_trace_task(device.trace_tasks + offset);
    const float2 result = __ldcs((float2*) (device.ptrs.trace_results + offset));
    const int pixel     = task.index.y * device.width + task.index.x;

    float depth     = result.x;
    uint32_t hit_id = __float_as_uint(result.y);

    RGBF record = load_RGBF(device.records + pixel);

    if (device.iteration_type == TYPE_LIGHT) {
      if (device.scene.fog.active) {
        const VolumeDescriptor volume = volume_get_descriptor_preset_fog();
        const float2 path             = volume_compute_path(volume, task.origin, task.ray, depth);

        if (path.x >= 0.0f) {
          record = scale_color(record, expf(-path.y * volume.max_scattering));
        }
      }

      if (device.scene.ocean.active) {
        const VolumeDescriptor volume = volume_get_descriptor_preset_ocean();
        const float2 path             = volume_compute_path(volume, task.origin, task.ray, depth);

        if (path.x >= 0.0f) {
          RGBF volume_transmittance = volume_get_transmittance(volume);

          record.r *= expf(-path.y * volume_transmittance.r);
          record.g *= expf(-path.y * volume_transmittance.g);
          record.b *= expf(-path.y * volume_transmittance.b);
        }
      }
    }
    else {
      const float random = quasirandom_sequence_1D(QUASI_RANDOM_TARGET_VOLUME_DIST, pixel);

      if (device.scene.fog.active) {
        const VolumeDescriptor volume = volume_get_descriptor_preset_fog();
        const float2 path             = volume_compute_path(volume, task.origin, task.ray, depth);

        if (path.x >= 0.0f) {
          const float volume_dist = volume_sample_intersection(volume, task.origin, task.ray, path.x, path.y, random);

          if (volume_dist < depth) {
            depth  = volume_dist;
            hit_id = HIT_TYPE_VOLUME_FOG;
          }
        }
      }

      if (device.scene.ocean.active) {
        bool ocean_intersection_possible = true;
        if (task.origin.y < OCEAN_MIN_HEIGHT || task.origin.y > OCEAN_MAX_HEIGHT) {
          const float short_distance  = ocean_short_distance(task.origin, task.ray);
          ocean_intersection_possible = (short_distance != FLT_MAX) && (short_distance <= depth);
        }

        if (ocean_intersection_possible) {
          const float ocean_depth = ocean_intersection_distance(task.origin, task.ray, depth);

          if (ocean_depth < depth) {
            depth  = ocean_depth;
            hit_id = HIT_TYPE_OCEAN;
          }
        }

        const VolumeDescriptor volume = volume_get_descriptor_preset_ocean();
        const float2 path             = volume_compute_path(volume, task.origin, task.ray, depth);

        if (path.x >= 0.0f) {
          const float volume_dist = volume_sample_intersection(volume, task.origin, task.ray, path.x, path.y, random);

          float integration_depth = path.y;

          if (volume_dist < depth) {
            depth  = volume_dist;
            hit_id = HIT_TYPE_VOLUME_OCEAN;

            integration_depth = depth - path.x;

            const float sampling_pdf = volume.max_scattering * expf(-integration_depth * volume.max_scattering);
            const RGBF target_pdf    = get_color(
              volume.scattering.r * expf(-integration_depth * volume.scattering.r),
              volume.scattering.g * expf(-integration_depth * volume.scattering.g),
              volume.scattering.b * expf(-integration_depth * volume.scattering.b));

            record = mul_color(record, scale_color(target_pdf, 1.0f / sampling_pdf));
          }

          record.r *= expf(-integration_depth * volume.absorption.r);
          record.g *= expf(-integration_depth * volume.absorption.g);
          record.b *= expf(-integration_depth * volume.absorption.b);
        }
      }

      __stcs((float2*) (device.ptrs.trace_results + offset), make_float2(depth, __uint_as_float(hit_id)));
    }

    store_RGBF(device.records + pixel, record);
  }
}
