#include "hip/hip_runtime.h"
// OptiX translation unit setup
#include "optix_compile_defines.cuh"
//

#include "math.cuh"
#include "memory.cuh"
#include "ocean_utils.cuh"
#include "optix_include.cuh"
#include "utils.cuh"

// Kernels must be named __[SEMANTIC]__..., for example, __raygen__...
// This can be found under function name prefix in the programming guide

////////////////////////////////////////////////////////////////////
// GBufferMetaData
////////////////////////////////////////////////////////////////////

__device__ uint16_t optix_float_to_bfloat16(const float val) {
  return __float_as_uint(val) >> 16;
}

__device__ void optix_write_out_gbuffer_meta(const DeviceTask task, OptixRaytraceResult result) {
  if (device.state.sample_id != 0 || device.state.depth != 0)
    return;

  const uint32_t shift = device.settings.supersampling + 1;
  const uint32_t mask  = (1 << shift) - 1;

  if ((task.index.x & mask) || (task.index.y & mask))
    return;

  const uint16_t x  = task.index.x >> shift;
  const uint16_t y  = task.index.y >> shift;
  const uint32_t ld = device.settings.width >> shift;

  if (device.ocean.active) {
    if (task.origin.y < OCEAN_MIN_HEIGHT || task.origin.y > OCEAN_MAX_HEIGHT) {
      const float short_distance = ocean_short_distance(task.origin, task.ray);

      if (short_distance < result.depth) {
        result.handle.instance_id = HIT_TYPE_REJECT;
        result.depth              = short_distance;
      }
    }
  }

  uint16_t material_id = MATERIAL_ID_INVALID;
  uint32_t instance_id = HIT_TYPE_INVALID;

  if (result.handle.instance_id < HIT_TYPE_TRIANGLE_ID_LIMIT) {
    const uint32_t mesh_id = mesh_id_load(result.handle.instance_id);

    material_id = material_id_load(mesh_id, result.handle.tri_id);
    instance_id = result.handle.instance_id;
  }

  vec3 rel_hit_pos = get_vector(0.0f, 0.0f, 0.0f);
  if (result.depth < FLT_MAX) {
    rel_hit_pos = scale_vector(task.ray, result.depth);
  }

  GBufferMetaData meta_data;

  meta_data.depth              = result.depth;
  meta_data.instance_id        = instance_id;
  meta_data.material_id        = material_id;
  meta_data.rel_hit_x_bfloat16 = optix_float_to_bfloat16(rel_hit_pos.x);
  meta_data.rel_hit_y_bfloat16 = optix_float_to_bfloat16(rel_hit_pos.y);
  meta_data.rel_hit_z_bfloat16 = optix_float_to_bfloat16(rel_hit_pos.z);

  uint4 data;
  data.x = meta_data.instance_id;
  data.y = __float_as_uint(meta_data.depth);
  data.z = (((uint32_t) meta_data.rel_hit_y_bfloat16) << 16) | (meta_data.rel_hit_x_bfloat16);
  data.w = (((uint32_t) meta_data.material_id) << 16) | (meta_data.rel_hit_z_bfloat16);

  __stwt((uint4*) device.ptrs.gbuffer_meta + x + y * ld, data);
}

////////////////////////////////////////////////////////////////////
// Raytracing passes
////////////////////////////////////////////////////////////////////

__device__ void optix_raytrace_geometry(const DeviceTask task, OptixRaytraceResult& result) {
  OptixKernelFunctionGeometryTracePayload payload;
  payload.depth  = result.depth;
  payload.handle = result.handle;

  optixKernelFunctionGeometryTrace(
    device.optix_bvh, task.origin, task.ray, 0.0f, result.depth, 0.0f, OptixVisibilityMask(0xFFFF), OPTIX_RAY_FLAG_NONE,
    OPTIX_TRACE_STATUS_EXECUTE, payload);

  result.depth  = payload.depth;
  result.handle = payload.handle;
}

__device__ void optix_raytrace_particles(const DeviceTask task, OptixRaytraceResult& result) {
  OptixTraceStatus trace_status = OPTIX_TRACE_STATUS_EXECUTE;

  // Particles can not be hit by non delta path due to their negligible contribution
  if ((task.state & STATE_FLAG_DELTA_PATH) == 0 || !device.particles.active) {
    trace_status = OPTIX_TRACE_STATUS_ABORT;
  }

  const float time         = quasirandom_sequence_1D_base_float(QUASI_RANDOM_TARGET_CAMERA_TIME, task.index, device.state.sample_id, 0);
  const vec3 motion        = angles_to_direction(device.particles.direction_altitude, device.particles.direction_azimuth);
  const vec3 motion_offset = scale_vector(motion, time * device.particles.speed);

  const vec3 scaled_ray = scale_vector(task.ray, 1.0f / device.particles.scale);
  vec3 pos              = scale_vector(add_vector(task.origin, motion_offset), 1.0f / device.particles.scale);

  // Map our current point into the particle tiling coordinate system
  pos.x = pos.x - floorf(pos.x);
  pos.y = pos.y - floorf(pos.y);
  pos.z = pos.z - floorf(pos.z);

  OptixKernelFunctionParticleTracePayload payload;
  payload.depth       = FLT_MAX;
  payload.instance_id = HIT_TYPE_REJECT;

  optixKernelFunctionParticleTrace(
    device.optix_bvh_particles, pos, scaled_ray, 0.0f, result.depth, 0.0f, OptixVisibilityMask(0xFFFF), OPTIX_RAY_FLAG_NONE, trace_status,
    payload);

  if (payload.instance_id != HIT_TYPE_REJECT) {
    // Hit ID contains the triangle ID but we only store the actual particle / quad ID
    payload.instance_id = HIT_TYPE_PARTICLE_MIN + (payload.instance_id >> 1);

    result.handle = triangle_handle_get(payload.instance_id, 0);
    result.depth  = payload.depth;
  }
}

extern "C" __global__ void __raygen__optix() {
  HANDLE_DEVICE_ABORT();

  const uint16_t trace_task_count = device.ptrs.trace_counts[THREAD_ID];
  const uint16_t task_id          = TASK_ID;

  if (task_id >= trace_task_count)
    return;

  const uint32_t offset = get_task_address(task_id);
  const DeviceTask task = task_load(offset);

  OptixRaytraceResult result;
  result.handle = triangle_handle_get(HIT_TYPE_SKY, 0);
  result.depth  = FLT_MAX;

  optix_raytrace_geometry(task, result);
  optix_raytrace_particles(task, result);

  triangle_handle_store(result.handle, offset);
  trace_depth_store(result.depth, offset);

  optix_write_out_gbuffer_meta(task, result);
}
