#include "hip/hip_runtime.h"
// Functions work differently when executed from this kernel
// This emulates the old device.iteration_type == TYPE_LIGHT checks.
#define SHADING_KERNEL
#define OPTIX_KERNEL

#include "bsdf.cuh"
#include "direct_lighting.cuh"
#include "geometry_utils.cuh"
#include "ior_stack.cuh"
#include "math.cuh"
#include "memory.cuh"
#include "shading_kernel.cuh"
#include "utils.cuh"

extern "C" __global__ void __raygen__optix() {
  HANDLE_DEVICE_ABORT();

  const int task_count  = device.ptrs.task_counts[TASK_ADDRESS_OFFSET_GEOMETRY];
  const int task_offset = device.ptrs.task_offsets[TASK_ADDRESS_OFFSET_GEOMETRY];

  for (int i = 0; i < task_count; i++) {
    const uint32_t offset                = get_task_address(task_offset + i);
    DeviceTask task                      = task_load(offset);
    const TriangleHandle triangle_handle = triangle_handle_load(offset);
    const float depth                    = trace_depth_load(offset);
    const uint32_t pixel                 = get_pixel_id(task.index);

    task.origin = add_vector(task.origin, scale_vector(task.ray, depth));

    GBufferData data = geometry_generate_g_buffer(task, triangle_handle, pixel);

    // We have to clamp due to numerical precision issues in the microfacet models.
    data.roughness = fmaxf(data.roughness, BSDF_ROUGHNESS_CLAMP);

    ////////////////////////////////////////////////////////////////////
    // Light Ray Sampling
    ////////////////////////////////////////////////////////////////////

    RGBF accumulated_light = splat_color(0.0f);
    accumulated_light      = add_color(accumulated_light, direct_lighting_sun(data, task.index));
    accumulated_light      = add_color(accumulated_light, direct_lighting_geometry(data, task.index));
    accumulated_light      = add_color(accumulated_light, direct_lighting_ambient(data, task.index));

    const RGBF record = load_RGBF(device.ptrs.records + pixel);

    accumulated_light = mul_color(accumulated_light, record);

    write_beauty_buffer(accumulated_light, pixel, task.state);
  }
}
