#include "hip/hip_runtime.h"
// OptiX translation unit setup
#include "optix_compile_defines.cuh"
//

#include "bsdf.cuh"
#include "direct_lighting.cuh"
#include "directives.cuh"
#include "ior_stack.cuh"
#include "math.cuh"
#include "memory.cuh"
#include "utils.cuh"
#include "volume_utils.cuh"

extern "C" __global__ void __raygen__optix() {
  HANDLE_DEVICE_ABORT();

#ifdef OPTIX_ENABLE_GEOMETRY_DL
  if (LIGHTS_ARE_PRESENT == false)
    return;
#endif

  const uint32_t task_count = device.ptrs.trace_counts[THREAD_ID];
  const uint32_t task_id    = TASK_ID;

  if (task_id >= task_count)
    return;

  const uint32_t task_base_address      = task_get_base_address(task_id, TASK_STATE_BUFFER_INDEX_PRESORT);
  DeviceTask task                       = task_load(task_base_address);
  const DeviceTaskTrace trace           = task_trace_load(task_base_address);
  const DeviceTaskThroughput throughput = task_throughput_load(task_base_address);

  const VolumeType volume_type = VolumeType(task.volume_id);

  if (volume_should_do_direct_lighting(volume_type, task.state) == false)
    return;

  const VolumeDescriptor volume = volume_get_descriptor_preset(volume_type);

  MaterialContextVolume ctx = volume_get_context(task, volume, trace.depth);

  RGBF accumulated_light = get_color(0.0f, 0.0f, 0.0f);

  accumulated_light = add_color(accumulated_light, direct_lighting_geometry(ctx, task.index));

  accumulated_light = mul_color(accumulated_light, record_unpack(throughput.record));

  const uint32_t pixel = get_pixel_id(task.index);
  write_beauty_buffer_indirect(accumulated_light, pixel);
}
