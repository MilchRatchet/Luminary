#include "hip/hip_runtime.h"
// More ideas:
// - Add a ray flag for if MIS is disabled (would be set for bounce rays coming from ocean surface): STATE_FLAG_MIS_DISABLED

#define UTILS_NO_DEVICE_TABLE

// Functions work differently when executed from this kernel
// This emulates the old device.iteration_type == TYPE_LIGHT checks.
#define SHADING_KERNEL
#define OPTIX_KERNEL

#include "utils.h"

extern "C" static __constant__ DeviceConstantMemory device;

#include "bsdf.cuh"
#include "directives.cuh"
#include "geometry.cuh"
#include "ior_stack.cuh"
#include "light.cuh"
#include "math.cuh"
#include "memory.cuh"
#include "ris.cuh"
#include "utils.cuh"

#define MAX_COMPRESSABLE_COLOR (1.99999988079071044921875f)

__device__ void optix_compress_color(RGBF color, unsigned int& data0, unsigned int& data1) {
  uint32_t bits_r = (__float_as_uint(fminf(color.r + 1.0f, MAX_COMPRESSABLE_COLOR)) >> 2) & 0x1FFFFF;
  uint32_t bits_g = (__float_as_uint(fminf(color.g + 1.0f, MAX_COMPRESSABLE_COLOR)) >> 2) & 0x1FFFFF;
  uint32_t bits_b = (__float_as_uint(fminf(color.b + 1.0f, MAX_COMPRESSABLE_COLOR)) >> 2) & 0x1FFFFF;

  data0 = bits_r | (bits_g << 21);
  data1 = (bits_g >> 11) | (bits_b << 10);
}

__device__ RGBF optix_decompress_color(unsigned int data0, unsigned int data1) {
  uint32_t bits_r = data0 & 0x1FFFFF;
  uint32_t bits_g = (data0 >> 21) & 0x7FF | ((data1 & 0x3FF) << 11);
  uint32_t bits_b = (data1 >> 10) & 0x1FFFFF;

  RGBF color;
  color.r = __uint_as_float(0x3F800000u | (bits_r << 2)) - 1.0f;
  color.g = __uint_as_float(0x3F800000u | (bits_g << 2)) - 1.0f;
  color.b = __uint_as_float(0x3F800000u | (bits_b << 2)) - 1.0f;

  return color;
}

extern "C" __global__ void __raygen__optix() {
  // For each [Light Queueable Task]
  //  Get GBufferData
  //
  //  Apply bounce MIS to emission and apply
  //
  //  Init normalization constant (sum weights = 0, num weights = 0)
  //  For [number of light rays]
  //    Sample light source with RIS
  //    Sample direction towards light with LTC + solid angle
  //    Compute Visibility
  //    If (Visibility > 0)
  //      Compute light weight and accumulate in local color value
  //
  //  Apply normalization constant to accumulated local color value
  //  Apply MIS weight to
  //
  //  Sample BRDF ray
  //  Queue bounce ray and store normalization constant

  const int task_count  = device.ptrs.task_counts[THREAD_ID * TASK_ADDRESS_COUNT_STRIDE + TASK_ADDRESS_OFFSET_GEOMETRY];
  const int task_offset = device.ptrs.task_offsets[THREAD_ID * TASK_ADDRESS_OFFSET_STRIDE + TASK_ADDRESS_OFFSET_GEOMETRY];
  int trace_count       = device.ptrs.trace_counts[THREAD_ID];

  for (int i = 0; i < task_count; i++) {
    GeometryTask task = load_geometry_task(device.ptrs.trace_tasks + get_task_address(task_offset + i));
    const int pixel   = task.index.y * device.width + task.index.x;

    GBufferData data = geometry_generate_g_buffer(task, pixel);

    write_normal_buffer(data.normal, pixel);

    if (!material_is_mirror(data.roughness, data.metallic))
      write_albedo_buffer(opaque_color(data.albedo), pixel);

    const RGBF record = load_RGBF(device.ptrs.records + pixel);

    RGBF accumulated_light =
      (state_peek(pixel, STATE_FLAG_BOUNCE_LIGHTING)) ? mul_color(data.emission, record) : get_color(0.0f, 0.0f, 0.0f);

    for (int j = 0; j < device.restir.num_light_rays; j++) {
      const uint32_t light_id   = ris_sample_light(data, task.index);
      const TriangleLight light = load_triangle_light(device.scene.triangle_lights, light_id);
      float pdf, dist;
      RGBF light_color;
      const vec3 dir = light_sample_triangle(light, data, task.index, pdf, dist, light_color);

      // TODO: Add support for transparent pass light directions
      const bool is_transparent_pass = false;
      const float shift              = (is_transparent_pass) ? -eps : eps;
      const vec3 shifted_position    = add_vector(data.position, scale_vector(data.V, shift * get_length(data.position)));

      const float3 origin = make_float3(shifted_position.x, shifted_position.y, shifted_position.z);

      pdf *= 1.0f / device.scene.triangle_lights_count;

      const float3 ray = make_float3(dir.x, dir.y, dir.z);

      // TODO: Make sure to set this to an invalid value for non triangle lights
      unsigned int hit_id = light_id;

      // 21 bits for each color component.
      unsigned int alpha_data0, alpha_data1;
      optix_compress_color(get_color(1.0f, 1.0f, 1.0f), alpha_data0, alpha_data1);

      // Disable OMM opaque hits because we want to know if we hit something that is fully opaque so we can reject.
      optixTrace(
        device.optix_bvh, origin, ray, 0.0f, dist, 0.0f, OptixVisibilityMask(0xFFFF), OPTIX_RAY_FLAG_ENFORCE_ANYHIT, 0, 0, 0, hit_id,
        alpha_data0, alpha_data1);

      RGBF visibility = optix_decompress_color(alpha_data0, alpha_data1);

      accumulated_light =
        add_color(accumulated_light, scale_color(mul_color(light_color, visibility), 1.0f / device.restir.num_light_rays));
    }

    accumulated_light = mul_color(accumulated_light, record);

    write_beauty_buffer(accumulated_light, pixel);

    BSDFSampleInfo bounce_info;
    float bsdf_marginal;
    vec3 bounce_ray = bsdf_sample(data, task.index, bounce_info, bsdf_marginal);

    RGBF bounce_record = mul_color(record, bounce_info.weight);

    const float shift = (bounce_info.is_transparent_pass) ? -eps : eps;
    data.position     = add_vector(data.position, scale_vector(data.V, shift * get_length(data.position)));

    if (bounce_info.is_transparent_pass) {
      const IORStackMethod ior_stack_method = (data.flags & G_BUFFER_REFRACTION_IS_INSIDE) ? IOR_STACK_METHOD_PULL : IOR_STACK_METHOD_PUSH;
      ior_stack_interact(data.ior_in, pixel, ior_stack_method);
    }

    TraceTask bounce_task;
    bounce_task.origin = data.position;
    bounce_task.ray    = bounce_ray;
    bounce_task.index  = task.index;

    if (validate_trace_task(bounce_task, bounce_record)) {
      store_RGBF(device.ptrs.records + pixel, bounce_record);
      store_trace_task(device.ptrs.trace_tasks + get_task_address(trace_count++), bounce_task);

      state_release(pixel, STATE_FLAG_BOUNCE_LIGHTING);
    }
  }

  device.ptrs.trace_counts[THREAD_ID] = trace_count;
}

/*
 * Performs alpha test on triangle
 * @result 0 if opaque, 1 if transparent, 2 if alpha cutoff
 */
__device__ RGBAF optix_alpha_test() {
  const unsigned int hit_id = optixGetPrimitiveIndex();

  const uint32_t material_id = load_triangle_material_id(hit_id);
  const uint16_t tex         = __ldg(&(device.scene.materials[material_id].albedo_map));

  RGBAF albedo = get_RGBAF(0.0f, 0.0f, 0.0f, 1.0f);

  if (tex != TEXTURE_NONE) {
    const UV uv = load_triangle_tex_coords(hit_id, optixGetTriangleBarycentrics());

    const float4 tex_value = tex2D<float4>(device.ptrs.albedo_atlas[tex].tex, uv.u, 1.0f - uv.v);

    albedo.r = tex_value.x;
    albedo.g = tex_value.y;
    albedo.b = tex_value.z;
    albedo.a = tex_value.w;
  }

  return albedo;
}

extern "C" __global__ void __anyhit__optix() {
  if (optixGetPrimitiveIndex() == optixGetPayload_0()) {
    optixIgnoreIntersection();
  }

  RGBAF albedo = optix_alpha_test();

  if (albedo.a == 0.0f) {
    optixIgnoreIntersection();
  }

  if (albedo.a == 1.0f) {
    optixSetPayload_0(HIT_TYPE_REJECT);

    optixTerminateRay();
  }

  RGBF alpha = (device.scene.material.colored_transparency) ? scale_color(opaque_color(albedo), 1.0f - albedo.a)
                                                            : get_color(1.0f - albedo.a, 1.0f - albedo.a, 1.0f - albedo.a);

  unsigned int alpha_data0 = optixGetPayload_1();
  unsigned int alpha_data1 = optixGetPayload_2();

  RGBF accumulated_alpha = optix_decompress_color(alpha_data0, alpha_data1);
  accumulated_alpha      = mul_color(accumulated_alpha, alpha);
  optix_compress_color(accumulated_alpha, alpha_data0, alpha_data1);

  optixSetPayload_1(alpha_data0);
  optixSetPayload_2(alpha_data1);

  optixIgnoreIntersection();
}

extern "C" __global__ void __closesthit__optix() {
  // Dummy closest hit, this will never get executed anyway due to the anyhit.
  // I could maybe get rid of this by adding more logic during the optix kernel compilation.
}
