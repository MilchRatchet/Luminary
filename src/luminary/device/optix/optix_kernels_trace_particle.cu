#include "hip/hip_runtime.h"
#define OPTIX_KERNEL

#define OPTIX_PAYLOAD_DEPTH 0
#define OPTIX_PAYLOAD_INSTANCE_ID 1

#include "math.cuh"
#include "memory.cuh"
#include "optix_utils.cuh"
#include "utils.cuh"

__device__ bool particle_opacity_cutout(const float2 coord) {
  const float dx = fabsf(coord.x - 0.5f);
  const float dy = fabsf(coord.y - 0.5f);

  const float r = dx * dx + dy * dy;

  return (r > 0.25f);
}

// Kernels must be named __[SEMANTIC]__..., for example, __raygen__...
// This can be found under function name prefix in the programming guide

extern "C" __global__ void __raygen__optix() {
  const uint16_t trace_task_count = device.ptrs.trace_counts[THREAD_ID];

  const float time         = quasirandom_sequence_1D_global(QUASI_RANDOM_TARGET_CAMERA_TIME);
  const vec3 motion        = angles_to_direction(device.particles.direction_altitude, device.particles.direction_azimuth);
  const vec3 motion_offset = scale_vector(motion, time * device.particles.speed);

  for (uint32_t i = 0; i < trace_task_count; i++) {
    const uint32_t offset    = get_task_address(i);
    const TraceTask task     = load_trace_task(offset);
    const TraceResult result = load_trace_result(offset);

    const vec3 scaled_ray = scale_vector(task.ray, 1.0f / device.particles.scale);
    const vec3 reference  = scale_vector(add_vector(task.origin, motion_offset), 1.0f / device.particles.scale);

    const float3 origin = make_float3(reference.x, reference.y, reference.z);
    const float3 ray    = make_float3(scaled_ray.x, scaled_ray.y, scaled_ray.z);

    float tmax = result.depth;

    unsigned int depth       = __float_as_uint(result.depth);
    unsigned int instance_id = result.instance_id;

    const unsigned int vis_mask = OptixVisibilityMask(0xFFFF);

    float t = 64.0f * eps;

    float3 inv_ray;
    inv_ray.x = 1.0f / ((fabsf(ray.x) > eps) ? ray.x : copysignf(eps, ray.x));
    inv_ray.y = 1.0f / ((fabsf(ray.y) > eps) ? ray.y : copysignf(eps, ray.y));
    inv_ray.z = 1.0f / ((fabsf(ray.z) > eps) ? ray.z : copysignf(eps, ray.z));

    for (int i = 0; i < 8; i++) {
      float3 p = make_float3(origin.x + ray.x * t, origin.y + ray.y * t, origin.z + ray.z * t);

      // Map our current point into the particle tiling coordinate system
      p.x = p.x - floorf(p.x);
      p.y = p.y - floorf(p.y);
      p.z = p.z - floorf(p.z);

      OPTIX_PAYLOAD_INDEX_REQUIRE(OPTIX_PAYLOAD_DEPTH, 0);
      OPTIX_PAYLOAD_INDEX_REQUIRE(OPTIX_PAYLOAD_INSTANCE_ID, 1);
      optixTrace(device.optix_bvh_particles, p, ray, 0.0f, tmax, 0.0f, vis_mask, OPTIX_RAY_FLAG_NONE, 0, 0, 0, depth, instance_id);

      const float intersection_dist = __uint_as_float(depth);

      if (intersection_dist < tmax) {
        // Hit ID contains the triangle ID but we only store the actual particle / quad ID
        instance_id = HIT_TYPE_PARTICLE_MIN + (instance_id >> 1);

        TraceResult trace_result;
        trace_result.depth       = t + intersection_dist;
        trace_result.instance_id = instance_id;
        trace_result.tri_id      = 0;

        store_trace_result(trace_result, offset);
        break;
      }
      else {
        const float tx = inv_ray.x * (((ray.x < 0.0f) ? 0.0f : 1.0f) - p.x);
        const float ty = inv_ray.y * (((ray.y < 0.0f) ? 0.0f : 1.0f) - p.y);
        const float tz = inv_ray.z * (((ray.z < 0.0f) ? 0.0f : 1.0f) - p.z);

        const float step = fminf(fminf(tx, ty), tz) + 128.0f * eps;

        t += step;
        tmax -= step;

        if (tmax < 0.0f)
          break;
      }
    }
  }
}

extern "C" __global__ void __anyhit__optix() {
  if (particle_opacity_cutout(optixGetTriangleBarycentrics())) {
    optixIgnoreIntersection();
  }
}

extern "C" __global__ void __closesthit__optix() {
  optixSetPayloadGeneric(OPTIX_PAYLOAD_DEPTH, __float_as_uint(optixGetRayTmax()));
  optixSetPayloadGeneric(OPTIX_PAYLOAD_INSTANCE_ID, optixGetPrimitiveIndex());
}
