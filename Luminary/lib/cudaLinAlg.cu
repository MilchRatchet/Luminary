#include "primitives.h"
#include <hip/hip_runtime_api.h>

__device__
vec3 cross_product(const vec3 a, const vec3 b) {
    vec3 result;

    result.x = a.y*b.z - a.z*b.y;
    result.y = a.z*b.x - a.x*b.z;
    result.z = a.x*b.y - a.y*b.x;

    return result;
}

__device__
float dot_product(const vec3 a, const vec3 b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__
vec3 vec_diff(const vec3 a, const vec3 b) {
    vec3 result;

    result.x = a.x - b.x;
    result.y = a.y - b.y;
    result.z = a.z - b.z;

    return result;
}
