#include "hip/hip_runtime.h"
// OptiX translation unit setup
#include "optix_compile_defines.cuh"
//

#include "bsdf.cuh"
#include "direct_lighting.cuh"
#include "directives.cuh"
#include "ior_stack.cuh"
#include "math.cuh"
#include "memory.cuh"
#include "utils.cuh"
#include "volume_utils.cuh"

extern "C" __global__ void __raygen__optix() {
  HANDLE_DEVICE_ABORT();

  const uint32_t task_count  = device.ptrs.task_counts[TASK_ADDRESS_OFFSET_VOLUME];
  const uint32_t task_offset = device.ptrs.task_offsets[TASK_ADDRESS_OFFSET_VOLUME];
  const uint32_t task_id     = TASK_ID;

  if (task_id >= task_count)
    return;

  const uint32_t offset       = get_task_address(task_offset + task_id);
  DeviceTask task             = task_load(offset);
  const TriangleHandle handle = triangle_handle_load(offset);
  const float depth           = trace_depth_load(offset);
  const uint32_t pixel        = get_pixel_id(task.index);

  task.origin = add_vector(task.origin, scale_vector(task.ray, depth));

  const VolumeType volume_type  = VOLUME_HIT_TYPE(handle.instance_id);
  const VolumeDescriptor volume = volume_get_descriptor_preset(volume_type);

  GBufferData data = volume_generate_g_buffer(task, handle.instance_id, pixel, volume);

  RGBF accumulated_light = get_color(0.0f, 0.0f, 0.0f);

  accumulated_light = add_color(accumulated_light, direct_lighting_geometry_bridges(task, volume_type, volume));
  accumulated_light = add_color(accumulated_light, direct_lighting_sun_phase(data, task.index));
  accumulated_light = add_color(accumulated_light, direct_lighting_ambient(data, task.index));

  const RGBF record = load_RGBF(device.ptrs.records + pixel);

  accumulated_light = mul_color(accumulated_light, record);

  write_beauty_buffer_indirect(accumulated_light, pixel);
}
