#include "denoiser.h"
#include <hip/hip_runtime.h>
#include <optix.h>
#include <optix_stubs.h>
#include <optix_function_table_definition.h>


#define OPTIX_CHECK( call )                                             \
  {                                                                     \
    OptixResult res = call;                                             \
    if( res != OPTIX_SUCCESS )                                          \
      {                                                                 \
        fprintf( stderr, "Optix returned error %d in call (%s) (line %d)\n", res, #call, __LINE__ ); \
        system("pause");                                                \
        exit(-1);                                                       \
      }                                                                 \
  }

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
  inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
  {
     if (code != hipSuccess)
     {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) {
          system("pause");
          exit(code);
        }
     }
  }

extern "C" void denoise_with_optix(raytrace_instance* instance) {

  OPTIX_CHECK(optixInit());

  OptixDeviceContext ctx;
  OPTIX_CHECK(optixDeviceContextCreate((hipCtx_t)0,(OptixDeviceContextOptions*)0, &ctx));

  OptixDenoiser denoiser;

  OptixDenoiserOptions opt;
  opt.inputKind = OPTIX_DENOISER_INPUT_RGB_ALBEDO;

  OPTIX_CHECK(optixDenoiserCreate(ctx, &opt, &denoiser));
  OPTIX_CHECK(optixDenoiserSetModel(denoiser, OPTIX_DENOISER_MODEL_KIND_HDR, nullptr, 0));

  OptixDenoiserSizes denoiserReturnSizes;
  OPTIX_CHECK(optixDenoiserComputeMemoryResources(denoiser, instance->width, instance->height, &denoiserReturnSizes));

  hipDeviceptr_t denoiserState;
  gpuErrchk(hipMalloc((void**) &denoiserState, denoiserReturnSizes.stateSizeInBytes));

  const size_t scratchSize = (denoiserReturnSizes.withoutOverlapScratchSizeInBytes > denoiserReturnSizes.withOverlapScratchSizeInBytes) ?
                          denoiserReturnSizes.withoutOverlapScratchSizeInBytes :
                          denoiserReturnSizes.withOverlapScratchSizeInBytes;

  hipDeviceptr_t denoiserScratch;
  gpuErrchk(hipMalloc((void**) &denoiserScratch, scratchSize));


  OPTIX_CHECK(optixDenoiserSetup(denoiser, 0,
    instance->width, instance->height,
    denoiserState,
    denoiserReturnSizes.stateSizeInBytes,
    denoiserScratch,
    scratchSize));

  gpuErrchk(hipMemcpy(instance->frame_output_gpu, instance->frame_output, sizeof(RGBF) * instance->width * instance->height, hipMemcpyHostToDevice));

  OptixImage2D inputLayer[2];

  inputLayer[0].data = (hipDeviceptr_t)instance->frame_output_gpu;
  inputLayer[0].width = instance->width;
  inputLayer[0].height = instance->height;
  inputLayer[0].rowStrideInBytes = instance->width * sizeof(RGBF);
  inputLayer[0].pixelStrideInBytes = sizeof(RGBF);
  inputLayer[0].format = OPTIX_PIXEL_FORMAT_FLOAT3;

  inputLayer[1].data = (hipDeviceptr_t)instance->albedo_buffer_gpu;
  inputLayer[1].width = instance->width;
  inputLayer[1].height = instance->height;
  inputLayer[1].rowStrideInBytes = instance->width * sizeof(RGBF);
  inputLayer[1].pixelStrideInBytes = sizeof(RGBF);
  inputLayer[1].format = OPTIX_PIXEL_FORMAT_FLOAT3;

  RGBF* output;
  gpuErrchk(hipMalloc((void**) &output, sizeof(RGBF) * instance->width * instance->height));

  OptixImage2D outputLayer;

  outputLayer.data = (hipDeviceptr_t)output;
  outputLayer.width = instance->width;
  outputLayer.height = instance->height;
  outputLayer.rowStrideInBytes = instance->width * sizeof(RGBF);
  outputLayer.pixelStrideInBytes = sizeof(RGBF);
  outputLayer.format = OPTIX_PIXEL_FORMAT_FLOAT3;

  hipDeviceptr_t hdr_intensity;
  gpuErrchk(hipMalloc((void**) &hdr_intensity, sizeof(float)));

  OPTIX_CHECK(optixDenoiserComputeIntensity(denoiser, 0, &inputLayer[0], hdr_intensity, denoiserScratch, scratchSize));

  hipDeviceptr_t avg_color;
  gpuErrchk(hipMalloc((void**) &avg_color, sizeof(float) * 3));

  OPTIX_CHECK(optixDenoiserComputeAverageColor(denoiser, 0, &inputLayer[0], avg_color, denoiserScratch, scratchSize));

  OptixDenoiserParams denoiserParams;
  denoiserParams.denoiseAlpha = 0;
  denoiserParams.hdrIntensity = hdr_intensity;
  denoiserParams.blendFactor = 0.0f;
  denoiserParams.hdrAverageColor = avg_color;

  OPTIX_CHECK(optixDenoiserInvoke(denoiser,
    0,
    &denoiserParams,
    denoiserState,
    denoiserReturnSizes.stateSizeInBytes,
    &inputLayer[0],
    2,
    0,
    0,
    &outputLayer,
    denoiserScratch,
    scratchSize));


  gpuErrchk(hipMemcpy(instance->frame_output, output, sizeof(RGBF) * instance->width * instance->height, hipMemcpyDeviceToHost));

  OPTIX_CHECK(optixDeviceContextDestroy(ctx));
  OPTIX_CHECK(optixDenoiserDestroy(denoiser));

  gpuErrchk(hipFree(output));
  gpuErrchk(hipFree((void*)hdr_intensity));
  gpuErrchk(hipFree((void*)avg_color));
  gpuErrchk(hipFree((void*)denoiserState));
  gpuErrchk(hipFree((void*)denoiserScratch));
}
