#include "hip/hip_runtime.h"
#include "scene.h"
#include "primitives.h"
#include "image.h"
#include "raytrace.h"
#include "mesh.h"
#include "SDL/SDL.h"
#include "cuda/utils.cuh"
#include "cuda/math.cuh"
#include "cuda/sky.cuh"
#include "cuda/brdf.cuh"
#include "cuda/bvh.cuh"
#include "cuda/directives.cuh"
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <optix.h>
#include <optix_stubs.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <chrono>
#include <thread>
#include <immintrin.h>

const static int threads_per_block = 128;
const static int blocks_per_grid = 512;

struct Sample {
    vec3 origin;
    vec3 ray;
    RGBF record;
    RGBF result;
    int depth;
    int state; //1st bit (finished?) 2nd bit (albedo buffer written?)
    int buffer_index;
} typedef Sample;

//---------------------------------
// Path Tracing
//---------------------------------

__device__
float get_light_angle(Light light, vec3 pos) {
    const float d = get_length(vec_diff(pos, light.pos)) + eps;
    return fminf(PI/2.0f,asinf(light.radius / d));
}

__device__
Sample trace_ray_iterative(const Sample input_sample, hiprandStateXORWOW_t* __restrict__ random) {
    int albedo_buffer_written = input_sample.state & 0b10;
    const int buffer_index = input_sample.buffer_index;

    RGBF result = input_sample.result;
    RGBF record = input_sample.record;

    vec3 ray = input_sample.ray;
    vec3 origin = input_sample.origin;

    int reflection_number;
    int state = input_sample.state;
    const int starting_threads = __popc(__activemask());

    for (reflection_number = input_sample.depth; reflection_number < device_reflection_depth; reflection_number++) {
        traversal_result traversal = traverse_bvh(origin, ray, device_scene.nodes, device_scene.traversal_triangles);

        if (traversal.hit_id == 0xffffffff) {
            RGBF sky = get_sky_color(ray);

            if (device_denoiser && !albedo_buffer_written) {
                RGBF sum = device_albedo_buffer[buffer_index];
                sum.r += sky.r;
                sum.g += sky.g;
                sum.b += sky.b;
                device_albedo_buffer[buffer_index] = sum;

                albedo_buffer_written++;
                state |= 0b10;
            }

            result.r += sky.r * record.r;
            result.g += sky.g * record.g;
            result.b += sky.b * record.b;

            state |= 0b1;

            break;
        }

        vec3 curr;
        curr.x = origin.x + ray.x * traversal.depth;
        curr.y = origin.y + ray.y * traversal.depth;
        curr.z = origin.z + ray.z * traversal.depth;

        const float4* hit_address = (float4*)(device_scene.triangles + traversal.hit_id);

        const float4 t1 = __ldg(hit_address);
        const float4 t2 = __ldg(hit_address + 1);
        const float4 t3 = __ldg(hit_address + 2);
        const float4 t4 = __ldg(hit_address + 3);
        const float4 t5 = __ldg(hit_address + 4);
        const float4 t6 = __ldg(hit_address + 5);
        const float4 t7 = __ldg(hit_address + 6);

        vec3 vertex;
        vertex.x = t1.x;
        vertex.y = t1.y;
        vertex.z = t1.z;

        vec3 edge1;
        edge1.x = t1.w;
        edge1.y = t2.x;
        edge1.z = t2.y;

        vec3 edge2;
        edge2.x = t2.z;
        edge2.y = t2.w;
        edge2.z = t3.x;

        vec3 normal = get_coordinates_in_triangle(vertex, edge1, edge2, curr);

        const float lambda = normal.x;
        const float mu = normal.y;

        vec3 vertex_normal;
        vertex_normal.x = t3.y;
        vertex_normal.y = t3.z;
        vertex_normal.z = t3.w;

        vec3 edge1_normal;
        edge1_normal.x = t4.x;
        edge1_normal.y = t4.y;
        edge1_normal.z = t4.z;

        vec3 edge2_normal;
        edge2_normal.x = t4.w;
        edge2_normal.y = t5.x;
        edge2_normal.z = t5.y;

        normal = lerp_normals(vertex_normal, edge1_normal, edge2_normal, lambda, mu);

        UV vertex_texture;
        vertex_texture.u = t5.z;
        vertex_texture.v = t5.w;

        UV edge1_texture;
        edge1_texture.u = t6.x;
        edge1_texture.v = t6.y;

        UV edge2_texture;
        edge2_texture.u = t6.z;
        edge2_texture.v = t6.w;

        const UV tex_coords = lerp_uv(vertex_texture, edge1_texture, edge2_texture, lambda, mu);

        vec3 face_normal;
        face_normal.x = t7.x;
        face_normal.y = t7.y;
        face_normal.z = t7.z;

        const int texture_object = __float_as_int(t7.w);

        const ushort4 maps = __ldg((ushort4*)(device_texture_assignments + texture_object));

        float roughness;
        float metallic;
        float intensity;

        if (maps.z) {
            const float4 material_f = tex2D<float4>(device_material_atlas[maps.z], tex_coords.u, 1.0f - tex_coords.v);

            roughness = (1.0f - material_f.x) * (1.0f - material_f.x);
            metallic = material_f.y;
            intensity = material_f.z * 255.0f;
        } else {
            roughness = 0.81f;
            metallic = 0.0f;
            intensity = 1.0f;
        }

        if (maps.y) {
            #ifdef LIGHTS_AT_NIGHT_ONLY
            if (device_sun.y < NIGHT_THRESHOLD) {
            #endif
            const float4 illuminance_f = tex2D<float4>(device_illuminance_atlas[maps.y], tex_coords.u, 1.0f - tex_coords.v);

            RGBF emission;
            emission.r = illuminance_f.x;
            emission.g = illuminance_f.y;
            emission.b = illuminance_f.z;

            result.r += emission.r * intensity * record.r;
            result.g += emission.g * intensity * record.g;
            result.b += emission.b * intensity * record.b;

            #ifdef FIRST_LIGHT_ONLY
            const double max_result = fmaxf(result.r, fmaxf(result.g, result.b));
            if (max_result > eps) {
                state |= 0b1;
                break;
            }
            #endif

            #ifdef LIGHTS_AT_NIGHT_ONLY
            }
            #endif
        }

        RGBAF albedo;

        if (maps.x) {
            const float4 albedo_f = tex2D<float4>(device_albedo_atlas[maps.x], tex_coords.u, 1.0f - tex_coords.v);
            albedo.r = albedo_f.x;
            albedo.g = albedo_f.y;
            albedo.b = albedo_f.z;
            albedo.a = albedo_f.w;
        } else {
            albedo.r = 0.9f;
            albedo.g = 0.9f;
            albedo.b = 0.9f;
            albedo.a = 1.0f;
        }

        if (hiprand_uniform(random) > albedo.a) {
            origin.x = curr.x + 2.0f * eps * ray.x;
            origin.y = curr.y + 2.0f * eps * ray.y;
            origin.z = curr.z + 2.0f * eps * ray.z;
        } else {
            if (device_denoiser && !albedo_buffer_written) {
                RGBF sum = device_albedo_buffer[buffer_index];
                sum.r += albedo.r;
                sum.g += albedo.g;
                sum.b += albedo.b;
                device_albedo_buffer[buffer_index] = sum;

                albedo_buffer_written++;
                state |= 0b10;
            }

            const float specular_probability = lerp(0.5f, 1.0f - eps, metallic);

            if (dot_product(normal, face_normal) < 0.0f) {
                face_normal = scale_vector(face_normal, -1.0f);
            }

            const vec3 V = scale_vector(ray, -1.0f);

            if (dot_product(face_normal, V) < 0.0f) {
                normal = scale_vector(normal, -1.0f);
                face_normal = scale_vector(face_normal, -1.0f);
            }

            origin.x = curr.x + face_normal.x * (eps * 8.0f);
            origin.y = curr.y + face_normal.y * (eps * 8.0f);
            origin.z = curr.z + face_normal.z * (eps * 8.0f);

            const float light_sample = hiprand_uniform(random);
            float light_angle;
            vec3 light_source;
            #ifdef LIGHTS_AT_NIGHT_ONLY
            const int light_count = (device_sun.y < NIGHT_THRESHOLD) ? device_scene.lights_length - 1 : 1;
            #else
            const int light_count = device_scene.lights_length;
            #endif


            if (light_sample < 0.5f) {
                #ifdef LIGHTS_AT_NIGHT_ONLY
                    const uint32_t light = (device_sun.y < NIGHT_THRESHOLD && light_count > 0) ? 1 + (uint32_t)(hiprand_uniform(random) * light_count) : 0;
                #else
                    const uint32_t light = (uint32_t)(hiprand_uniform(random) * light_count);
                #endif

                const float4 light_data = __ldg((float4*)(device_scene.lights + light));
                vec3 light_pos;
                light_pos.x = light_data.x;
                light_pos.y = light_data.y;
                light_pos.z = light_data.z;
                light_pos = vec_diff(light_pos, origin);
                light_source = normalize_vector(light_pos);
                const float d = get_length(light_pos) + eps;
                light_angle = fminf(PI/2.0f,asinf(light_data.w / d)) * 2.0f / PI;
            }

            if (hiprand_uniform(random) < specular_probability) {
                const float alpha = roughness * roughness;

                const float beta = acosf(hiprand_uniform(random));
                const float gamma = 2.0f * 3.1415926535f * hiprand_uniform(random);

                const Quaternion rotation_to_z = get_rotation_to_z_canonical(normal);

                float weight = 1.0f;

                const vec3 V_local = rotate_vector_by_quaternion(V, rotation_to_z);
                vec3 H_local;

                if (alpha < eps) {
                    H_local.x = 0.0f;
                    H_local.y = 0.0f;
                    H_local.z = 1.0f;
                } else {
                    const vec3 S_local = rotate_vector_by_quaternion(
                        normalize_vector(sample_ray_from_angles_and_vector(beta * light_angle, gamma, light_source)),
                        rotation_to_z);

                    if (light_sample < 0.5f && S_local.z > 0.0f) {
                        H_local.x = S_local.x + V_local.x;
                        H_local.y = S_local.y + V_local.y;
                        H_local.z = S_local.z + V_local.z;

                        H_local = normalize_vector(H_local);

                        weight = 2.0f * light_angle * light_count;
                    } else {
                        H_local = sample_GGX_VNDF(V_local, alpha, hiprand_uniform(random), hiprand_uniform(random));

                        if (S_local.z > 0.0f) weight = 2.0f;
                    }
                }

                const vec3 ray_local = reflect_vector(scale_vector(V_local, -1.0f), H_local);

                const float HdotR = fmaxf(eps, fminf(1.0f, dot_product(H_local, ray_local)));
                const float NdotR = fmaxf(eps, fminf(1.0f, ray_local.z));
                const float NdotV = fmaxf(eps, fminf(1.0f, V_local.z));

                ray = normalize_vector(rotate_vector_by_quaternion(ray_local, inverse_quaternion(rotation_to_z)));

                vec3 specular_f0;
                specular_f0.x = lerp(0.04f, albedo.r, metallic);
                specular_f0.y = lerp(0.04f, albedo.g, metallic);
                specular_f0.z = lerp(0.04f, albedo.b, metallic);

                const vec3 F = Fresnel_Schlick(specular_f0, shadowed_F90(specular_f0), HdotR);

                const float milchs_energy_recovery = lerp(1.0f, 1.51f + 1.51f * NdotV, roughness);

                weight *= milchs_energy_recovery * Smith_G2_over_G1_height_correlated(alpha * alpha, NdotR, NdotV) / specular_probability;

                record.r *= F.x * weight;
                record.g *= F.y * weight;
                record.b *= F.z * weight;
            }
            else
            {
                float weight = 1.0f;

                const float alpha = acosf(sqrtf(hiprand_uniform(random)));
                const float gamma = 2.0f * PI * hiprand_uniform(random);

                ray = normalize_vector(sample_ray_from_angles_and_vector(alpha * light_angle, gamma, light_source));
                const float light_feasible = dot_product(ray, normal);

                if (light_sample < 0.5f && light_feasible >= 0.0f) {
                    weight = 2.0f * light_angle * light_count;
                } else {
                    ray = sample_ray_from_angles_and_vector(alpha, gamma, normal);

                    if (light_feasible >=0.0f) weight = 2.0f;
                }

                vec3 H;
                H.x = V.x + ray.x;
                H.y = V.y + ray.y;
                H.z = V.z + ray.z;
                H = normalize_vector(H);

                const float half_angle = fmaxf(eps, fminf(dot_product(H,ray),1.0f));
                const float energyFactor = lerp(1.0f, 1.0f/1.51f, roughness);

                const float FD90MinusOne = 0.5f * roughness + 2.0f * half_angle * half_angle * roughness - 1.0f;

                const float angle = fmaxf(eps, fminf(dot_product(normal, ray),1.0f));
                const float previous_angle = fmaxf(eps, fminf(dot_product(V, normal),1.0f));

                const float FDL = 1.0f + (FD90MinusOne * __powf(1.0f - angle, 5.0f));
                const float FDV = 1.0f + (FD90MinusOne * __powf(1.0f - previous_angle, 5.0f));

                weight *= FDL * FDV * energyFactor * (1.0f - metallic) / (1.0f - specular_probability);

                record.r *= albedo.r * weight;
                record.g *= albedo.g * weight;
                record.b *= albedo.b * weight;
            }
        }



        #ifdef WEIGHT_BASED_EXIT
        const double max_record = fmaxf(record.r, fmaxf(record.g, record.b));
        if (max_record < CUTOFF ||
        (max_record < PROBABILISTIC_CUTOFF && hiprand_uniform(random) > (max_record - CUTOFF)/(CUTOFF-PROBABILISTIC_CUTOFF)))
        {
            state |= 0b1;
            break;
        }
        #endif

        #ifdef LOW_QUALITY_LONG_BOUNCES
        if (reflection_number >= MIN_BOUNCES && hiprand_uniform(random) < 1.0f/device_reflection_depth) {
            state |= 0b1;
            break;
        }
        #endif

        if (__popc(__activemask()) < 0.0f *  starting_threads) break;
    }

    if (reflection_number >= device_reflection_depth - 1) state |= 0b1;

    Sample output_sample;
    output_sample.origin = origin;
    output_sample.ray = ray;
    output_sample.result = result;
    output_sample.record = record;
    output_sample.state = state;
    output_sample.depth = reflection_number;
    output_sample.buffer_index = buffer_index;

    return output_sample;
}

__global__
void trace_rays(volatile uint32_t* progress, int offset_x, int offset_y, int size_x, int size_y, int limit) {
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;

    hiprandStateXORWOW_t random;

    hiprand_init(id + clock(), 0, 0, &random);
    Sample sample;
    sample.state = 0b11;
    int i = 0;

    RGBF pixel;
    pixel.r = 0.0f;
    pixel.g = 0.0f;
    pixel.b = 0.0f;

    while (id < device_amount && id < limit) {
        if (sample.state & 0b1) {
            vec3 ray;
            vec3 default_ray;

            const int x = offset_x + (id / 8) % size_x;
            const int y = offset_y + (id % 8) + 8 * (id / (8 * size_x));

            default_ray.x = -device_scene.camera.fov + device_step * x + device_offset_x * hiprand_uniform(&random) * 2.0f;
            default_ray.y = device_vfov - device_step * y - device_offset_y * hiprand_uniform(&random) * 2.0f;
            default_ray.z = -1.0f;

            ray = normalize_vector(rotate_vector_by_quaternion(default_ray, device_camera_rotation));

            sample.ray = ray;
            sample.origin = device_scene.camera.pos;
            sample.buffer_index = x + y * device_width;
            sample.state = 0;
            sample.depth = 0;
            sample.record.r = 1.0f;
            sample.record.g = 1.0f;
            sample.record.b = 1.0f;
            sample.result.r = 0.0f;
            sample.result.g = 0.0f;
            sample.result.b = 0.0f;
        }

        sample = trace_ray_iterative(sample, &random);

        if (sample.state & 0b1) {
            if (isnan(sample.result.r) || isinf(sample.result.r)) {
                sample.result.r = 1.0f;
            }
            if (isnan(sample.result.g) || isinf(sample.result.g)) {
                sample.result.g = 0.0f;
            }
            if (isnan(sample.result.b) || isinf(sample.result.b)) {
                sample.result.b = 0.0f;
            }

            pixel.r += sample.result.r;
            pixel.g += sample.result.g;
            pixel.b += sample.result.b;

            i++;
        }

        if (i == device_diffuse_samples) {
            const float weight = 1.0f/(float)device_diffuse_samples;

            pixel.r *= weight;
            pixel.g *= weight;
            pixel.b *= weight;

            const int x = offset_x + (id / 8) % size_x;
            const int y = offset_y + (id % 8) + 8 * (id / (8 * size_x));

            const int buffer_index = x + y * device_width;

            if (device_denoiser) {
                RGBF sum = device_albedo_buffer[buffer_index];
                sum.r *= weight;
                sum.g *= weight;
                sum.b *= weight;
                device_albedo_buffer[buffer_index] = sum;
            }

            device_frame[buffer_index] = pixel;

            id += blockDim.x * gridDim.x;
            i = 0;

            pixel.r = 0.0f;
            pixel.g = 0.0f;
            pixel.b = 0.0f;

            atomicAdd((uint32_t*)progress, 1);
            __threadfence_system();
        }
    }
}

__global__
void set_up_raytracing_device() {
    hiprand_init(0,0,0,&device_random);
}

static void update_sun(const Scene scene) {
    vec3 sun;
    sun.x = sinf(scene.azimuth) * cosf(scene.altitude);
    sun.y = sinf(scene.altitude);
    sun.z = cosf(scene.azimuth) * cosf(scene.altitude);
    sun = normalize_vector(sun);

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_sun), &(sun), sizeof(vec3), 0, hipMemcpyHostToDevice));

    const vec3 light_source_sun = scale_vector(sun, 149630000000.0f);

    gpuErrchk(hipMemcpy(scene.lights, &light_source_sun, sizeof(vec3), hipMemcpyHostToDevice));
}

static void update_camera_pos(const Scene scene, const unsigned int width, const unsigned int height) {
    const float alpha = scene.camera.rotation.x;
    const float beta = scene.camera.rotation.y;
    const float gamma = scene.camera.rotation.z;

    const float cy = cosf(gamma * 0.5f);
    const float sy = sinf(gamma * 0.5f);
    const float cp = cosf(beta * 0.5f);
    const float sp = sinf(beta * 0.5f);
    const float cr = cosf(alpha * 0.5f);
    const float sr = sinf(alpha * 0.5f);

    Quaternion q;
    q.w = cr * cp * cy + sr * sp * sy;
    q.x = sr * cp * cy - cr * sp * sy;
    q.y = cr * sp * cy + sr * cp * sy;
    q.z = cr * cp * sy - sr * sp * cy;

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_camera_rotation), &(q), sizeof(Quaternion), 0, hipMemcpyHostToDevice));

    const float step = 2.0f * (scene.camera.fov / width);
    const float vfov = step * height / 2.0f;
    const float offset_x = (step / 2.0f);
    const float offset_y = (step / 2.0f);

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_step), &(step), sizeof(float), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_vfov), &(vfov), sizeof(float), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_offset_x), &(offset_x), sizeof(float), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_offset_y), &(offset_y), sizeof(float), 0, hipMemcpyHostToDevice));
}

extern "C" raytrace_instance* init_raytracing(
    const unsigned int width, const unsigned int height, const int reflection_depth,
    const int diffuse_samples, void* albedo_atlas, int albedo_atlas_length, void* illuminance_atlas,
    int illuminance_atlas_length, void* material_atlas, int material_atlas_length, Scene scene, int denoiser) {

    raytrace_instance* instance = (raytrace_instance*)malloc(sizeof(raytrace_instance));

    instance->width = width;
    instance->height = height;
    instance->frame_buffer = (RGBF*)_mm_malloc(sizeof(RGBF) * width * height, 32);

    const unsigned int amount = width * height;
    set_up_raytracing_device<<<1,1>>>();

    gpuErrchk(hipMalloc((void**) &(instance->frame_buffer_gpu), sizeof(RGBF) * width * height));

    instance->reflection_depth = reflection_depth;
    instance->diffuse_samples = diffuse_samples;

    instance->albedo_atlas = albedo_atlas;
    instance->illuminance_atlas = illuminance_atlas;
    instance->material_atlas = material_atlas;

    instance->albedo_atlas_length = albedo_atlas_length;
    instance->illuminance_atlas_length = illuminance_atlas_length;
    instance->material_atlas_length = material_atlas_length;

    instance->scene_gpu = scene;

    gpuErrchk(hipMalloc((void**) &(instance->scene_gpu.texture_assignments), sizeof(texture_assignment) * scene.materials_length));
    gpuErrchk(hipMalloc((void**) &(instance->scene_gpu.triangles), sizeof(Triangle) * instance->scene_gpu.triangles_length));
    gpuErrchk(hipMalloc((void**) &(instance->scene_gpu.traversal_triangles), sizeof(Traversal_Triangle) * instance->scene_gpu.triangles_length));
    gpuErrchk(hipMalloc((void**) &(instance->scene_gpu.nodes), sizeof(Node) * instance->scene_gpu.nodes_length));
    gpuErrchk(hipMalloc((void**) &(instance->scene_gpu.node_addresses), sizeof(int) * instance->scene_gpu.node_addresses_length));
    gpuErrchk(hipMalloc((void**) &(instance->scene_gpu.lights), sizeof(Light) * instance->scene_gpu.lights_length));

    gpuErrchk(hipMemcpy(instance->scene_gpu.texture_assignments, scene.texture_assignments, sizeof(texture_assignment) * scene.materials_length, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(instance->scene_gpu.triangles, scene.triangles, sizeof(Triangle) * scene.triangles_length, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(instance->scene_gpu.traversal_triangles, scene.traversal_triangles, sizeof(Traversal_Triangle) * scene.triangles_length, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(instance->scene_gpu.nodes, scene.nodes, sizeof(Node) * scene.nodes_length, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(instance->scene_gpu.node_addresses, scene.node_addresses, sizeof(int) * scene.node_addresses_length, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(instance->scene_gpu.lights, scene.lights, sizeof(Light) * scene.lights_length, hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_texture_assignments), &(instance->scene_gpu.texture_assignments), sizeof(texture_assignment*), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_frame), &(instance->frame_buffer_gpu), sizeof(RGBF*), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_width), &(instance->width), sizeof(unsigned int), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_height), &(instance->height), sizeof(unsigned int), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_reflection_depth), &(instance->reflection_depth), sizeof(int), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_diffuse_samples), &(instance->diffuse_samples), sizeof(int), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_albedo_atlas), &(instance->albedo_atlas), sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_illuminance_atlas), &(instance->illuminance_atlas), sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_material_atlas), &(instance->material_atlas), sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_amount), &(amount), sizeof(unsigned int), 0, hipMemcpyHostToDevice));

    instance->denoiser = denoiser;

    if (instance->denoiser) {
        gpuErrchk(hipMalloc((void**) &(instance->albedo_buffer_gpu), sizeof(RGBF) * width * height));
        gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_albedo_buffer), &(instance->albedo_buffer_gpu), sizeof(RGBF*), 0, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_denoiser), &(instance->denoiser), sizeof(int), 0, hipMemcpyHostToDevice));
    }

    return instance;
}

extern "C" void* initialize_textures(TextureRGBA* textures, const int textures_length) {
    hipTextureObject_t* textures_cpu = (hipTextureObject_t*) malloc(sizeof(hipTextureObject_t) * textures_length);
    hipTextureObject_t* textures_gpu;

    gpuErrchk(hipMalloc((void**) &(textures_gpu), sizeof(hipTextureObject_t) * textures_length));

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0]   = hipAddressModeWrap;
    texDesc.addressMode[1]   = hipAddressModeWrap;
    texDesc.filterMode       = hipFilterModeLinear;
    texDesc.readMode         = hipReadModeElementType;
    texDesc.normalizedCoords = 1;

    for (int i = 0; i < textures_length; i++) {
        TextureRGBA texture = textures[i];

        const int num_rows = texture.height;
        const int num_cols = texture.width;
        RGBAF* data = texture.data;
        RGBAF* data_gpu;
        size_t pitch;
        gpuErrchk(hipMallocPitch((void**) &data_gpu, &pitch, num_cols * sizeof(RGBAF), num_rows));
        gpuErrchk(hipMemcpy2D(data_gpu, pitch, data, num_cols * sizeof(RGBAF), num_cols * sizeof(RGBAF), num_rows, hipMemcpyHostToDevice));

        struct hipResourceDesc resDesc;
        memset(&resDesc, 0, sizeof(resDesc));
        resDesc.resType = hipResourceTypePitch2D;
        resDesc.res.pitch2D.devPtr = data_gpu;
        resDesc.res.pitch2D.width = num_cols;
        resDesc.res.pitch2D.height = num_rows;
        resDesc.res.pitch2D.desc = hipCreateChannelDesc<float4>();
        resDesc.res.pitch2D.pitchInBytes = pitch;

        gpuErrchk(hipCreateTextureObject(textures_cpu + i, &resDesc, &texDesc, NULL));
    }

    gpuErrchk(hipMemcpy(textures_gpu, textures_cpu, sizeof(hipTextureObject_t) * textures_length, hipMemcpyHostToDevice));

    free(textures_cpu);

    return textures_gpu;
}

extern "C" void initialize_device() {
    gpuErrchk(hipSetDeviceFlags(hipDeviceMapHost));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    printf("Luminary - %s\n",prop.name);
}

extern "C" void free_textures(void* texture_atlas, const int textures_length) {
    hipTextureObject_t* textures_cpu = (hipTextureObject_t*) malloc(sizeof(hipTextureObject_t) * textures_length);
    gpuErrchk(hipMemcpy(textures_cpu, texture_atlas, sizeof(hipTextureObject_t) * textures_length, hipMemcpyDeviceToHost));

    for (int i = 0; i < textures_length; i++) {
        gpuErrchk(hipDestroyTextureObject(textures_cpu[i]));
    }

    gpuErrchk(hipFree(texture_atlas));
    free(textures_cpu);
}

__device__ __host__
static float linearRGB_to_SRGB(const float value) {
    if (value <= 0.0031308f) {
      return 12.92f * value;
    }
    else {
      return 1.055f * powf(value, 0.416666666667f) - 0.055f;
    }
  }

extern "C" void copy_framebuffer_to_cpu(raytrace_instance* instance) {
    gpuErrchk(hipMemcpy(instance->frame_buffer, instance->frame_buffer_gpu, sizeof(RGBF) * instance->width * instance->height, hipMemcpyDeviceToHost));
}

extern "C" void trace_scene(Scene scene, raytrace_instance* instance, const int progress) {
    volatile uint32_t *progress_gpu, *progress_cpu;

    hipEvent_t kernelFinished;
    gpuErrchk(hipEventCreate(&kernelFinished));
    gpuErrchk(hipHostAlloc((void**)&progress_cpu, sizeof(uint32_t), hipHostMallocMapped));
    gpuErrchk(hipHostGetDevicePointer((uint32_t**)&progress_gpu, (uint32_t*)progress_cpu, 0));
    *progress_cpu = 0;

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_scene), &(instance->scene_gpu), sizeof(Scene), 0, hipMemcpyHostToDevice));

    update_sun(instance->scene_gpu);
    update_camera_pos(instance->scene_gpu, instance->width, instance->height);

    clock_t t = clock();

    if (progress == 2) {
        const unsigned int max_block_width = 128;
        const unsigned int max_block_height = 128;


        SDL_Init(SDL_INIT_VIDEO);
        SDL_Window* window = SDL_CreateWindow(
          "Luminary", SDL_WINDOWPOS_CENTERED, SDL_WINDOWPOS_CENTERED, instance->width, instance->height,
          SDL_WINDOW_SHOWN);

        SDL_Surface* window_surface = SDL_GetWindowSurface(window);

        Uint32 rmask, gmask, bmask, amask;

      #if SDL_BYTEORDER == SDL_BIG_ENDIAN
        rmask = 0xff000000;
        gmask = 0x00ff0000;
        bmask = 0x0000ff00;
        amask = 0x000000ff;
      #else
        rmask = 0x000000ff;
        gmask = 0x0000ff00;
        bmask = 0x00ff0000;
        amask = 0xff000000;
      #endif

        SDL_Surface* surface =
          SDL_CreateRGBSurface(0, instance->width, instance->height, 24, rmask, gmask, bmask, amask);

        RGB8* buffer = (RGB8*) surface->pixels;

        int exit = 0;

        char* title = (char*) malloc(4096);

        unsigned int offset_x = 0;
        unsigned int offset_y = 0;
        unsigned int block_width = 0;
        unsigned int block_height = 0;

        unsigned int block_counter = 0;
        const unsigned int blocks_per_row = (instance->width / max_block_width) + 1;
        const unsigned int total_blocks = blocks_per_row * ((instance->height / max_block_height) + 1);

        while (!exit && block_counter < total_blocks) {
          SDL_Event event;

          for (unsigned int i = 0; i < block_height; i++) {
            gpuErrchk(hipMemcpy(
                instance->frame_buffer + offset_x + (offset_y + i) * instance->width,
                instance->frame_buffer_gpu + offset_x + (offset_y + i) * instance->width,
                sizeof(RGBF) * block_width,
                hipMemcpyDeviceToHost));
          }

          const unsigned int block_x = block_counter % blocks_per_row;
          const unsigned int block_y = block_counter / blocks_per_row;
          const unsigned int new_offset_x = block_x * max_block_width;
          const unsigned int new_offset_y = block_y * max_block_height;

          const unsigned int new_block_width = (new_offset_x + max_block_width > instance->width) ? instance->width - new_offset_x : max_block_width;
          const unsigned int new_block_height = (new_offset_y + max_block_height > instance->height) ? instance->height - new_offset_y : max_block_height;

          trace_rays<<<blocks_per_grid,threads_per_block>>>(progress_gpu, new_offset_x , new_offset_y, new_block_width, new_block_height, new_block_width * new_block_height);

          for (unsigned int j = new_offset_y; j < new_offset_y + new_block_height; j++) {
            for (unsigned int i = new_offset_x; i < new_offset_x + new_block_width; i++) {
              RGB8 pixel;
              pixel.r = 255;
              pixel.g = 255;
              pixel.b = 255;

              buffer[i + instance->width * j] = pixel;
            }
          }

          for (unsigned int j = offset_y; j < offset_y + block_height; j++) {
              for (unsigned int i = offset_x; i < offset_x + block_width; i++) {
                RGB8 pixel;
                RGBF pixel_float = instance->frame_buffer[i + instance->width * j];

                RGBF color;
                color.r = min(255.9f, linearRGB_to_SRGB(pixel_float.r) * 255.9f);
                color.g = min(255.9f, linearRGB_to_SRGB(pixel_float.g) * 255.9f);
                color.b = min(255.9f, linearRGB_to_SRGB(pixel_float.b) * 255.9f);

                pixel.r = (uint8_t) color.r;
                pixel.g = (uint8_t) color.g;
                pixel.b = (uint8_t) color.b;

                buffer[i + instance->width * j] = pixel;
              }
          }

          SDL_BlitSurface(surface, 0, window_surface, 0);

          clock_t curr_time = clock();
          const double time_elapsed = (((double)curr_time - t)/CLOCKS_PER_SEC);
          sprintf(title, "Luminary - Progress: %2.1f%% - Time Elapsed: %.1fs - Time Remaining: %.1fs", (((float)block_counter) / total_blocks) * 100.0f, time_elapsed, (total_blocks - block_counter) * (time_elapsed/block_counter));

          SDL_SetWindowTitle(window, title);
          SDL_UpdateWindowSurface(window);

          while (SDL_PollEvent(&event)) {
            if (event.type == SDL_QUIT) {
              exit = 1;
            }
          }

          offset_x = new_offset_x;
          offset_y = new_offset_y;
          block_width = new_block_width;
          block_height = new_block_height;

          block_counter++;
        }

        free(title);

        SDL_DestroyWindow(window);
        SDL_Quit();
    } else if (progress == 1) {
        trace_rays<<<blocks_per_grid,threads_per_block>>>(progress_gpu, 0, 0, instance->width, instance->height, instance->width * instance->height);

        gpuErrchk(hipEventRecord(kernelFinished));

        uint32_t progress = 0;
        const uint32_t total_pixels = instance->width * instance->height;
        const float ratio = 1.0f/((float)instance->width * (float)instance->height);
        while (hipEventQuery(kernelFinished) != hipSuccess) {
            std::this_thread::sleep_for(std::chrono::microseconds(100000));
            gpuErrchk(hipPeekAtLastError());
            uint32_t new_progress = *progress_cpu;
            if (new_progress > progress) {
                progress = new_progress;
            }
            clock_t curr_time = clock();
            const double time_elapsed = (((double)curr_time - t)/CLOCKS_PER_SEC);
            printf("\r                                                                                                          \rProgress: %2.1f%% - Time Elapsed: %.1fs - Time Remaining: %.1fs - Performance: %.1f Mrays/s",
                (float)progress * ratio * 100, time_elapsed,
                (total_pixels - progress) * (time_elapsed/progress),
                0.000001 * instance->diffuse_samples * instance->reflection_depth * progress / time_elapsed);
        }

        printf("\r                                                                                                              \r");
    } else {
        trace_rays<<<blocks_per_grid,threads_per_block>>>(progress_gpu, 0, 0, instance->width, instance->height, instance->width * instance->height);
    }

    gpuErrchk(hipDeviceSynchronize());
}

extern "C" void free_inputs(raytrace_instance* instance) {
    gpuErrchk(hipFree(instance->scene_gpu.texture_assignments));
    gpuErrchk(hipFree(instance->scene_gpu.triangles));
    gpuErrchk(hipFree(instance->scene_gpu.traversal_triangles));
    gpuErrchk(hipFree(instance->scene_gpu.nodes));
    gpuErrchk(hipFree(instance->scene_gpu.node_addresses));
    gpuErrchk(hipFree(instance->scene_gpu.lights));
}

extern "C" void free_outputs(raytrace_instance* instance) {
    gpuErrchk(hipFree(instance->frame_buffer_gpu));
    _mm_free(instance->frame_buffer);

    if (instance->denoiser) {
        gpuErrchk(hipFree(instance->albedo_buffer_gpu));
    }

    free(instance);
}

__device__
RGB8* device_frame_8bit;

__device__
RGBF tonemap(RGBF pixel) {
  const float a = 2.51f;
  const float b = 0.03f;
  const float c = 2.43f;
  const float d = 0.59f;
  const float e = 0.14f;

  pixel.r = 1.25f * (pixel.r * (a * pixel.r + b)) / (pixel.r * (c * pixel.r + d) + e);
  pixel.g = 1.25f * (pixel.g * (a * pixel.g + b)) / (pixel.g * (c * pixel.g + d) + e);
  pixel.b = 1.25f * (pixel.b * (a * pixel.b + b)) / (pixel.b * (c * pixel.b + d) + e);

  return pixel;
}

__global__
void convert_RGBF_to_RGB8(const RGBF* source) {
  unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;

  const unsigned int amount = device_width * device_height;

  while (id < amount) {
    int x = id % device_width;
    int y = id / device_width;

    RGBF pixel = source[x + y * device_width];

    pixel = tonemap(pixel);

    pixel.r = fminf(255.9f, 255.9f * linearRGB_to_SRGB(pixel.r));
    pixel.g = fminf(255.9f, 255.9f * linearRGB_to_SRGB(pixel.g));
    pixel.b = fminf(255.9f, 255.9f * linearRGB_to_SRGB(pixel.b));

    RGB8 converted_pixel;
    converted_pixel.r = (uint8_t)pixel.r;
    converted_pixel.g = (uint8_t)pixel.g;
    converted_pixel.b = (uint8_t)pixel.b;

    device_frame_8bit[x + y * device_width] = converted_pixel;

    id += blockDim.x * gridDim.x;
  }
}

extern "C" void initiliaze_realtime(raytrace_instance* instance) {
    gpuErrchk(hipMalloc((void**) &(instance->buffer_8bit_gpu), sizeof(RGB8) * instance->width * instance->height));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_frame_8bit), &(instance->buffer_8bit_gpu), sizeof(RGB8*), 0, hipMemcpyHostToDevice));
}

extern "C" void free_realtime(raytrace_instance* instance) {
    gpuErrchk(hipFree(instance->buffer_8bit_gpu));
}

extern "C" void copy_framebuffer_to_8bit(RGB8* buffer, RGBF* source, raytrace_instance* instance) {
    convert_RGBF_to_RGB8<<<blocks_per_grid,threads_per_block>>>(source);
    gpuErrchk(hipMemcpy(buffer, instance->buffer_8bit_gpu, sizeof(RGB8) * instance->width * instance->height, hipMemcpyDeviceToHost));
}

struct realtime_denoise {
    OptixDeviceContext ctx;
    OptixDenoiser denoiser;
    OptixDenoiserOptions opt;
    OptixDenoiserSizes denoiserReturnSizes;
    hipDeviceptr_t denoiserState;
    hipDeviceptr_t denoiserScratch;
    OptixImage2D inputLayer[2];
    OptixImage2D outputLayer;
    hipDeviceptr_t hdr_intensity;
    hipDeviceptr_t avg_color;
} typedef realtime_denoise;

extern "C" void* initialize_optix_denoise_for_realtime(raytrace_instance* instance) {
    OPTIX_CHECK(optixInit());

    realtime_denoise* denoise_setup = (realtime_denoise*)malloc(sizeof(realtime_denoise));

    OPTIX_CHECK(optixDeviceContextCreate((hipCtx_t)0,(OptixDeviceContextOptions*)0, &denoise_setup->ctx));

    denoise_setup->opt.inputKind = OPTIX_DENOISER_INPUT_RGB_ALBEDO;

    OPTIX_CHECK(optixDenoiserCreate(denoise_setup->ctx, &denoise_setup->opt, &denoise_setup->denoiser));
    OPTIX_CHECK(optixDenoiserSetModel(denoise_setup->denoiser, OPTIX_DENOISER_MODEL_KIND_HDR, nullptr, 0));

    OPTIX_CHECK(optixDenoiserComputeMemoryResources(denoise_setup->denoiser, instance->width, instance->height, &denoise_setup->denoiserReturnSizes));

    gpuErrchk(hipMalloc((void**) &denoise_setup->denoiserState, denoise_setup->denoiserReturnSizes.stateSizeInBytes));

    const size_t scratchSize = (denoise_setup->denoiserReturnSizes.withoutOverlapScratchSizeInBytes > denoise_setup->denoiserReturnSizes.withOverlapScratchSizeInBytes) ?
                                denoise_setup->denoiserReturnSizes.withoutOverlapScratchSizeInBytes :
                                denoise_setup->denoiserReturnSizes.withOverlapScratchSizeInBytes;

    gpuErrchk(hipMalloc((void**) &denoise_setup->denoiserScratch, scratchSize));


    OPTIX_CHECK(optixDenoiserSetup(denoise_setup->denoiser, 0,
      instance->width, instance->height,
      denoise_setup->denoiserState,
      denoise_setup->denoiserReturnSizes.stateSizeInBytes,
      denoise_setup->denoiserScratch,
      scratchSize));

    denoise_setup->inputLayer[0].data = (hipDeviceptr_t)instance->frame_buffer_gpu;
    denoise_setup->inputLayer[0].width = instance->width;
    denoise_setup->inputLayer[0].height = instance->height;
    denoise_setup->inputLayer[0].rowStrideInBytes = instance->width * sizeof(RGBF);
    denoise_setup->inputLayer[0].pixelStrideInBytes = sizeof(RGBF);
    denoise_setup->inputLayer[0].format = OPTIX_PIXEL_FORMAT_FLOAT3;

    denoise_setup->inputLayer[1].data = (hipDeviceptr_t)instance->albedo_buffer_gpu;
    denoise_setup->inputLayer[1].width = instance->width;
    denoise_setup->inputLayer[1].height = instance->height;
    denoise_setup->inputLayer[1].rowStrideInBytes = instance->width * sizeof(RGBF);
    denoise_setup->inputLayer[1].pixelStrideInBytes = sizeof(RGBF);
    denoise_setup->inputLayer[1].format = OPTIX_PIXEL_FORMAT_FLOAT3;

    RGBF* output;
    gpuErrchk(hipMalloc((void**) &output, sizeof(RGBF) * instance->width * instance->height));

    denoise_setup->outputLayer.data = (hipDeviceptr_t)output;
    denoise_setup->outputLayer.width = instance->width;
    denoise_setup->outputLayer.height = instance->height;
    denoise_setup->outputLayer.rowStrideInBytes = instance->width * sizeof(RGBF);
    denoise_setup->outputLayer.pixelStrideInBytes = sizeof(RGBF);
    denoise_setup->outputLayer.format = OPTIX_PIXEL_FORMAT_FLOAT3;

    gpuErrchk(hipMalloc((void**) &denoise_setup->hdr_intensity, sizeof(float)));

    gpuErrchk(hipMalloc((void**) &denoise_setup->avg_color, sizeof(float) * 3));

    return denoise_setup;
}

extern "C" RGBF* denoise_with_optix_realtime(void* input) {
    realtime_denoise* denoise_setup = (realtime_denoise*) input;

    const size_t scratchSize = (denoise_setup->denoiserReturnSizes.withoutOverlapScratchSizeInBytes > denoise_setup->denoiserReturnSizes.withOverlapScratchSizeInBytes) ?
                                denoise_setup->denoiserReturnSizes.withoutOverlapScratchSizeInBytes :
                                denoise_setup->denoiserReturnSizes.withOverlapScratchSizeInBytes;

    OPTIX_CHECK(optixDenoiserComputeIntensity(denoise_setup->denoiser, 0, &denoise_setup->inputLayer[0], denoise_setup->hdr_intensity, denoise_setup->denoiserScratch, scratchSize));

    OPTIX_CHECK(optixDenoiserComputeAverageColor(denoise_setup->denoiser, 0, &denoise_setup->inputLayer[0], denoise_setup->avg_color, denoise_setup->denoiserScratch, scratchSize));

    OptixDenoiserParams denoiserParams;
    denoiserParams.denoiseAlpha = 0;
    denoiserParams.hdrIntensity = denoise_setup->hdr_intensity;
    denoiserParams.blendFactor = 0.0f;
    denoiserParams.hdrAverageColor = denoise_setup->avg_color;

    OPTIX_CHECK(optixDenoiserInvoke(denoise_setup->denoiser,
        0,
        &denoiserParams,
        denoise_setup->denoiserState,
        denoise_setup->denoiserReturnSizes.stateSizeInBytes,
        &denoise_setup->inputLayer[0],
        2,
        0,
        0,
        &denoise_setup->outputLayer,
        denoise_setup->denoiserScratch,
        scratchSize));

    return (RGBF*)denoise_setup->outputLayer.data;
}

extern "C" void free_realtime_denoise(void* input) {
    realtime_denoise denoise_setup = *(realtime_denoise*) input;

    OPTIX_CHECK(optixDeviceContextDestroy(denoise_setup.ctx));
    OPTIX_CHECK(optixDenoiserDestroy(denoise_setup.denoiser));

    gpuErrchk(hipFree((void*)denoise_setup.outputLayer.data));
    gpuErrchk(hipFree((void*)denoise_setup.hdr_intensity));
    gpuErrchk(hipFree((void*)denoise_setup.avg_color));
    gpuErrchk(hipFree((void*)denoise_setup.denoiserState));
    gpuErrchk(hipFree((void*)denoise_setup.denoiserScratch));
}
