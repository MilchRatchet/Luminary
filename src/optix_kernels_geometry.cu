#include "hip/hip_runtime.h"
#define UTILS_NO_DEVICE_TABLE

// Functions work differently when executed from this kernel
// This emulates the old device.iteration_type == TYPE_LIGHT checks.
#define SHADING_KERNEL
#define OPTIX_KERNEL

#include "utils.h"

extern "C" static __constant__ DeviceConstantMemory device;

#include "bsdf.cuh"
#include "directives.cuh"
#include "geometry_utils.cuh"
#include "ior_stack.cuh"
#include "math.cuh"
#include "memory.cuh"
#include "shading_kernel.cuh"
#include "toy_utils.cuh"
#include "utils.cuh"

extern "C" __global__ void __raygen__optix() {
  const int task_count  = device.ptrs.task_counts[THREAD_ID * TASK_ADDRESS_COUNT_STRIDE + TASK_ADDRESS_OFFSET_GEOMETRY];
  const int task_offset = device.ptrs.task_offsets[THREAD_ID * TASK_ADDRESS_OFFSET_STRIDE + TASK_ADDRESS_OFFSET_GEOMETRY];
  int trace_count       = device.ptrs.trace_counts[THREAD_ID];

  for (int i = 0; i < task_count; i++) {
    const ShadingTask task = load_shading_task(device.ptrs.trace_tasks + get_task_address(task_offset + i));
    const int pixel        = task.index.y * device.width + task.index.x;

    GBufferData data;
    if (task.hit_id == HIT_TYPE_TOY) {
      data = toy_generate_g_buffer(task, pixel);
    }
    else if (task.hit_id == HIT_TYPE_OCEAN) {
      data = ocean_generate_g_buffer(task, pixel);
    }
    else {
      data = geometry_generate_g_buffer(task, pixel);
    }

    write_normal_buffer(data.normal, pixel);

    if (!material_is_mirror(data.roughness, data.metallic))
      write_albedo_buffer(opaque_color(data.albedo), pixel);

    const bool is_delta_path = state_peek(pixel, STATE_FLAG_DELTA_PATH);

    ////////////////////////////////////////////////////////////////////
    // Bounce Ray Sampling
    ////////////////////////////////////////////////////////////////////

    BSDFSampleInfo bounce_info;
    vec3 bounce_ray = bsdf_sample(data, task.index, bounce_info);

    ////////////////////////////////////////////////////////////////////
    // Update delta path state
    ////////////////////////////////////////////////////////////////////

    bool is_delta_distribution;
    if (bounce_info.is_transparent_pass) {
      const float refraction_scale = (data.ior_in > data.ior_out) ? data.ior_in / data.ior_out : data.ior_out / data.ior_in;
      is_delta_distribution        = data.roughness * fminf(refraction_scale - 1.0f, 1.0f) <= GEOMETRY_DELTA_PATH_CUTOFF;
    }
    else {
      is_delta_distribution = bounce_info.is_microfacet_based && (data.roughness <= GEOMETRY_DELTA_PATH_CUTOFF);
    }

    const bool is_pass_through = bsdf_is_pass_through_ray(bounce_info.is_transparent_pass, data.ior_in, data.ior_out);

    ////////////////////////////////////////////////////////////////////
    // Light Ray Sampling
    ////////////////////////////////////////////////////////////////////

    // We clamp the roughness to avoid caustics which would never clean up.
    if (!is_delta_path) {
      data.roughness = fmaxf(data.roughness, device.scene.material.caustic_roughness_clamp);
    }

    bool use_light_rays = true;
    if (task.hit_id == HIT_TYPE_OCEAN && !is_delta_path) {
      use_light_rays = false;
    }

    RGBF accumulated_light = (state_peek(pixel, STATE_FLAG_CAMERA_DIRECTION)) ? data.emission : get_color(0.0f, 0.0f, 0.0f);

    if (use_light_rays) {
      accumulated_light = add_color(accumulated_light, optix_compute_light_ray_sun(data, task.index));
      accumulated_light = add_color(accumulated_light, optix_compute_light_ray_toy(data, task.index));
      accumulated_light = add_color(accumulated_light, optix_compute_light_ray_geo(data, task.index));
    }

    accumulated_light = add_color(
      accumulated_light,
      optix_compute_light_ray_ambient_sky(data, bounce_ray, bounce_info.weight, bounce_info.is_transparent_pass, task.index));

    const RGBF record = load_RGBF(device.ptrs.records + pixel);

    accumulated_light = mul_color(accumulated_light, record);

    write_beauty_buffer(accumulated_light, pixel);

    if (bounce_info.is_transparent_pass) {
      const IORStackMethod ior_stack_method = (data.flags & G_BUFFER_REFRACTION_IS_INSIDE) ? IOR_STACK_METHOD_PULL : IOR_STACK_METHOD_PUSH;
      ior_stack_interact(data.ior_out, pixel, ior_stack_method);
    }

    data.position = shift_origin_vector(data.position, data.V, bounce_ray, bounce_info.is_transparent_pass);

    TraceTask bounce_task;
    bounce_task.origin = data.position;
    bounce_task.ray    = bounce_ray;
    bounce_task.index  = task.index;

    RGBF bounce_record = mul_color(record, bounce_info.weight);

    // This must be done after the trace rays due to some optimization in the compiler.
    // The compiler reloads these values at some point for some reason and if we overwrite
    // the values we will get garbage. I am not sure if this is a compiler bug or some undefined
    // behaviour on my side.
    if (validate_trace_task(bounce_task, bounce_record)) {
      store_trace_task(device.ptrs.trace_tasks + get_task_address(trace_count++), bounce_task);
      store_RGBF(device.ptrs.records + pixel, bounce_record);

      uint32_t flags_to_release = 0;

      if (is_delta_path && !is_delta_distribution) {
        flags_to_release |= STATE_FLAG_DELTA_PATH;
      }

      if (!is_pass_through) {
        flags_to_release |= STATE_FLAG_CAMERA_DIRECTION;
      }

      state_release(pixel, flags_to_release);
    }
  }

  device.ptrs.trace_counts[THREAD_ID] = trace_count;
}
