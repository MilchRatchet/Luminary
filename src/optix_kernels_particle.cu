#include "hip/hip_runtime.h"
#define UTILS_NO_DEVICE_TABLE

#include "utils.h"

extern "C" static __constant__ DeviceConstantMemory device;

#include "math.cuh"
#include "memory.cuh"
#include "utils.cuh"

__device__ bool particle_opacity_cutout(const float2 coord) {
  const float dx = fabsf(coord.x - 0.5f);
  const float dy = fabsf(coord.y - 0.5f);

  const float r = dx * dx + dy * dy;

  return (r > 0.25f);
}

// Kernels must be named __[SEMANTIC]__..., for example, __raygen__...
// This can be found under function name prefix in the programming guide

extern "C" __global__ void __raygen__optix() {
  const uint3 idx  = optixGetLaunchIndex();
  const uint3 dimx = optixGetLaunchDimensions();

  const uint16_t trace_task_count = device.ptrs.trace_counts[idx.x + idx.y * dimx.x];

  const float time         = quasirandom_sequence_1D_global(QUASI_RANDOM_TARGET_CAMERA_TIME);
  const vec3 motion        = angles_to_direction(device.scene.particles.direction_altitude, device.scene.particles.direction_azimuth);
  const vec3 motion_offset = scale_vector(motion, time * device.scene.particles.speed);

  for (int i = 0; i < trace_task_count; i++) {
    const int offset     = get_task_address2(idx.x, idx.y, i);
    const TraceTask task = load_trace_task_essentials(device.ptrs.trace_tasks + offset);
    const float2 result  = __ldcs((float2*) (device.ptrs.trace_results + offset));

    const vec3 scaled_ray = scale_vector(task.ray, 1.0f / device.scene.particles.scale);
    const vec3 reference  = scale_vector(add_vector(task.origin, motion_offset), 1.0f / device.scene.particles.scale);

    const float3 origin = make_float3(reference.x, reference.y, reference.z);
    const float3 ray    = make_float3(scaled_ray.x, scaled_ray.y, scaled_ray.z);

    float tmax = result.x;

    unsigned int depth  = __float_as_uint(result.x);
    unsigned int hit_id = __float_as_uint(result.y);
    unsigned int cost   = 0;

    const unsigned int vis_mask = OptixVisibilityMask(0xFFFF);

    float t = 64.0f * eps;

    float3 inv_ray;
    inv_ray.x = 1.0f / ((fabsf(ray.x) > eps) ? ray.x : copysignf(eps, ray.x));
    inv_ray.y = 1.0f / ((fabsf(ray.y) > eps) ? ray.y : copysignf(eps, ray.y));
    inv_ray.z = 1.0f / ((fabsf(ray.z) > eps) ? ray.z : copysignf(eps, ray.z));

    for (int i = 0; i < 8; i++) {
      float3 p = make_float3(origin.x + ray.x * t, origin.y + ray.y * t, origin.z + ray.z * t);

      // Map our current point into the particle tiling coordinate system
      p.x = p.x - floorf(p.x);
      p.y = p.y - floorf(p.y);
      p.z = p.z - floorf(p.z);

      optixTrace(device.optix_bvh_particles, p, ray, 0.0f, tmax, 0.0f, vis_mask, OPTIX_RAY_FLAG_NONE, 0, 0, 0, depth, hit_id, cost);

      const float intersection_dist = __uint_as_float(depth);

      if (intersection_dist < tmax) {
        float2 trace_result = result;

        // Hit ID contains the triangle ID but we only store the actual particle / quad ID
        hit_id = HIT_TYPE_PARTICLE_MIN + (hit_id >> 1);

        if (device.shading_mode == SHADING_HEAT) {
          trace_result = make_float2(cost, __uint_as_float(hit_id));
        }
        else {
          trace_result = make_float2(t + intersection_dist, __uint_as_float(hit_id));
        }

        __stcs((float2*) (device.ptrs.trace_results + offset), trace_result);
        break;
      }
      else {
        const float tx = inv_ray.x * (((ray.x < 0.0f) ? 0.0f : 1.0f) - p.x);
        const float ty = inv_ray.y * (((ray.y < 0.0f) ? 0.0f : 1.0f) - p.y);
        const float tz = inv_ray.z * (((ray.z < 0.0f) ? 0.0f : 1.0f) - p.z);

        const float step = fminf(fminf(tx, ty), tz) + 128.0f * eps;

        t += step;
        tmax -= step;

        if (tmax < 0.0f)
          break;
      }
    }
  }
}

extern "C" __global__ void __anyhit__optix() {
  optixSetPayload_2(optixGetPayload_2() + 1);

  if (particle_opacity_cutout(optixGetTriangleBarycentrics())) {
    optixIgnoreIntersection();
  }
}

extern "C" __global__ void __closesthit__optix() {
  optixSetPayload_0(__float_as_uint(optixGetRayTmax()));
  optixSetPayload_1(optixGetPrimitiveIndex());
}
