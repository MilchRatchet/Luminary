#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cudatest.h"
#include <hip/hip_runtime.h>

__global__
void addcu(int a, int b, int* c) {
  *c = a + b;
}

extern "C" void cudatest() {
  int c;
  int* dev_c;
  hipMalloc((void**) &dev_c, sizeof(int));
  addcu<<<1, 1>>>(2, 7, dev_c);

  hipMemcpy(&c,dev_c,sizeof(int),hipMemcpyDeviceToHost);

  printf("2 + 7 = %d\n",c);

  hipFree(dev_c);
}


