#include "hip/hip_runtime.h"
// OptiX translation unit setup
#include "optix_compile_defines.cuh"
//

#include "bsdf.cuh"
#include "direct_lighting.cuh"
#include "geometry_utils.cuh"
#include "ior_stack.cuh"
#include "math.cuh"
#include "memory.cuh"
#include "utils.cuh"

extern "C" __global__ void __raygen__optix() {
  HANDLE_DEVICE_ABORT();

#ifdef OPTIX_ENABLE_GEOMETRY_DL
  if (LIGHTS_ARE_PRESENT == false)
    return;
#endif

  const uint32_t task_count  = device.ptrs.task_counts[TASK_ADDRESS_OFFSET_GEOMETRY];
  const uint32_t task_offset = device.ptrs.task_offsets[TASK_ADDRESS_OFFSET_GEOMETRY];
  const uint32_t task_id     = TASK_ID;

  if (task_id >= task_count)
    return;

  const uint32_t task_base_address = task_get_base_address(task_offset + task_id, TASK_STATE_BUFFER_INDEX_POSTSORT);
  DeviceTask task                  = task_load(task_base_address);
  const DeviceTaskTrace trace      = task_trace_load(task_base_address);

#ifdef OPTIX_ENABLE_GEOMETRY_DL
  if (direct_lighting_geometry_is_valid(task) == false)
    return;
#endif

  task.origin = add_vector(task.origin, scale_vector(task.ray, trace.depth));

  GeometryContextCreationInfo ctx_creation_info;
  ctx_creation_info.task  = task;
  ctx_creation_info.trace = trace;
  ctx_creation_info.hints = GEOMETRY_CONTEXT_CREATION_HINT_DL;

  const MaterialContextGeometry ctx = geometry_get_context(ctx_creation_info);

  ////////////////////////////////////////////////////////////////////
  // Light Ray Sampling
  ////////////////////////////////////////////////////////////////////

  RGBF accumulated_light = splat_color(0.0f);

#ifdef OPTIX_ENABLE_GEOMETRY_DL
  accumulated_light = add_color(accumulated_light, direct_lighting_geometry(ctx, task.index));
#endif

#ifdef OPTIX_ENABLE_SKY_DL
  accumulated_light = add_color(accumulated_light, direct_lighting_sun(ctx, task.index));
  accumulated_light = add_color(accumulated_light, direct_lighting_ambient(ctx, task.index));
#endif

  const DeviceTaskThroughput throughput = task_throughput_load(task_base_address);

  accumulated_light = mul_color(accumulated_light, record_unpack(throughput.record));

  const uint32_t pixel = get_pixel_id(task.index);
  write_beauty_buffer(accumulated_light, pixel, task.state);
}
