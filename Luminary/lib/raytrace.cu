#include "hip/hip_runtime.h"
#include "scene.h"
#include "primitives.h"
#include "image.h"
#include "raytrace.h"
#include "mesh.h"
#include "error.h"
#include "SDL/SDL.h"
#include "cuda/utils.cuh"
#include "cuda/math.cuh"
#include "cuda/sky.cuh"
#include "cuda/brdf.cuh"
#include "cuda/bvh.cuh"
#include "cuda/directives.cuh"
#include "cuda/random.cuh"
#include "cuda/kernels.cuh"
#include "cuda/denoise.cuh"
#include <hip/hip_runtime_api.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <chrono>
#include <thread>
#include <immintrin.h>

//---------------------------------
// Path Tracing
//---------------------------------

static void update_sun(const Scene scene) {
    vec3 sun;
    sun.x = sinf(scene.azimuth) * cosf(scene.altitude);
    sun.y = sinf(scene.altitude);
    sun.z = cosf(scene.azimuth) * cosf(scene.altitude);
    const float scale = 1.0f / (sqrtf(sun.x * sun.x + sun.y * sun.y + sun.z * sun.z));
    sun.x *= scale;
    sun.y *= scale;
    sun.z *= scale;

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_sun), &(sun), sizeof(vec3), 0, hipMemcpyHostToDevice));

    const vec3 light_source_sun = scale_vector(sun, 149630000000.0f);

    gpuErrchk(hipMemcpy(scene.lights, &light_source_sun, sizeof(vec3), hipMemcpyHostToDevice));
}

static void update_camera_pos(const Scene scene, const unsigned int width, const unsigned int height) {
    const float alpha = scene.camera.rotation.x;
    const float beta = scene.camera.rotation.y;
    const float gamma = scene.camera.rotation.z;

    const float cy = cosf(gamma * 0.5f);
    const float sy = sinf(gamma * 0.5f);
    const float cp = cosf(beta * 0.5f);
    const float sp = sinf(beta * 0.5f);
    const float cr = cosf(alpha * 0.5f);
    const float sr = sinf(alpha * 0.5f);

    Quaternion q;
    q.w = cr * cp * cy + sr * sp * sy;
    q.x = sr * cp * cy - cr * sp * sy;
    q.y = cr * sp * cy + sr * cp * sy;
    q.z = cr * cp * sy - sr * sp * cy;

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_camera_rotation), &(q), sizeof(Quaternion), 0, hipMemcpyHostToDevice));

    const float step = 2.0f * (scene.camera.fov / width);
    const float vfov = step * height / 2.0f;
    const float offset_x = (step / 2.0f);
    const float offset_y = (step / 2.0f);

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_step), &(step), sizeof(float), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_vfov), &(vfov), sizeof(float), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_offset_x), &(offset_x), sizeof(float), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_offset_y), &(offset_y), sizeof(float), 0, hipMemcpyHostToDevice));
}


extern "C" RaytraceInstance* init_raytracing(
    const unsigned int width, const unsigned int height, const int max_ray_depth, const int samples, void* albedo_atlas, int albedo_atlas_length, void* illuminance_atlas,
    int illuminance_atlas_length, void* material_atlas, int material_atlas_length, Scene scene, int denoiser) {

    RaytraceInstance* instance = (RaytraceInstance*)malloc(sizeof(RaytraceInstance));

    instance->width = width;
    instance->height = height;
    instance->frame_output = (RGBF*)_mm_malloc(sizeof(RGBF) * width * height, 32);

    const unsigned int amount = width * height;

    gpuErrchk(hipMalloc((void**) &(instance->frame_buffer_gpu), sizeof(RGBF) * width * height));
    gpuErrchk(hipMalloc((void**) &(instance->frame_output_gpu), sizeof(RGBF) * width * height));
    gpuErrchk(hipMalloc((void**) &(instance->frame_variance_gpu), sizeof(RGBF) * width * height));
    gpuErrchk(hipMalloc((void**) &(instance->frame_bias_cache_gpu), sizeof(RGBF) * width * height));
    gpuErrchk(hipMalloc((void**) &(instance->records_gpu), sizeof(RGBF) * width * height));

    instance->max_ray_depth = max_ray_depth;
    instance->offline_samples = samples;

    instance->albedo_atlas = albedo_atlas;
    instance->illuminance_atlas = illuminance_atlas;
    instance->material_atlas = material_atlas;

    instance->albedo_atlas_length = albedo_atlas_length;
    instance->illuminance_atlas_length = illuminance_atlas_length;
    instance->material_atlas_length = material_atlas_length;

    instance->default_material.r = 0.5f;
    instance->default_material.g = 0.0f;
    instance->default_material.b = 1.0f;

    instance->scene_gpu = scene;
    instance->shading_mode = 0;

    gpuErrchk(hipMalloc((void**) &(instance->scene_gpu.texture_assignments), sizeof(texture_assignment) * scene.materials_length));
    gpuErrchk(hipMalloc((void**) &(instance->scene_gpu.triangles), sizeof(Triangle) * instance->scene_gpu.triangles_length));
    gpuErrchk(hipMalloc((void**) &(instance->scene_gpu.traversal_triangles), sizeof(Traversal_Triangle) * instance->scene_gpu.triangles_length));
    gpuErrchk(hipMalloc((void**) &(instance->scene_gpu.nodes), sizeof(Node8) * instance->scene_gpu.nodes_length));
    gpuErrchk(hipMalloc((void**) &(instance->scene_gpu.lights), sizeof(Light) * instance->scene_gpu.lights_length));

    gpuErrchk(hipMemcpy(instance->scene_gpu.texture_assignments, scene.texture_assignments, sizeof(texture_assignment) * scene.materials_length, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(instance->scene_gpu.triangles, scene.triangles, sizeof(Triangle) * scene.triangles_length, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(instance->scene_gpu.traversal_triangles, scene.traversal_triangles, sizeof(Traversal_Triangle) * scene.triangles_length, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(instance->scene_gpu.nodes, scene.nodes, sizeof(Node8) * scene.nodes_length, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(instance->scene_gpu.lights, scene.lights, sizeof(Light) * scene.lights_length, hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_texture_assignments), &(instance->scene_gpu.texture_assignments), sizeof(texture_assignment*), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_frame_buffer), &(instance->frame_buffer_gpu), sizeof(RGBF*), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_frame_output), &(instance->frame_output_gpu), sizeof(RGBF*), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_frame_variance), &(instance->frame_variance_gpu), sizeof(RGBF*), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_frame_bias_cache), &(instance->frame_bias_cache_gpu), sizeof(RGBF*), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_records), &(instance->records_gpu), sizeof(RGBF*), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_width), &(instance->width), sizeof(unsigned int), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_height), &(instance->height), sizeof(unsigned int), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_max_ray_depth), &(instance->max_ray_depth), sizeof(int), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_albedo_atlas), &(instance->albedo_atlas), sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_illuminance_atlas), &(instance->illuminance_atlas), sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_material_atlas), &(instance->material_atlas), sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_amount), &(amount), sizeof(unsigned int), 0, hipMemcpyHostToDevice));

    instance->denoiser = denoiser;

    if (instance->denoiser) {
        gpuErrchk(hipMalloc((void**) &(instance->albedo_buffer_gpu), sizeof(RGBF) * width * height));
        gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_albedo_buffer), &(instance->albedo_buffer_gpu), sizeof(RGBF*), 0, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_denoiser), &(instance->denoiser), sizeof(int), 0, hipMemcpyHostToDevice));

        gpuErrchk(hipMalloc((void**) &(instance->bloom_scratch_gpu), sizeof(RGBF) * width * height));
        gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_bloom_scratch), &(instance->bloom_scratch_gpu), sizeof(RGBF*), 0, hipMemcpyHostToDevice));
    }

    const int thread_count = THREADS_PER_BLOCK * BLOCKS_PER_GRID;
    const int pixels_per_thread = (amount + thread_count - 1) / thread_count;
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_pixels_per_thread), &(pixels_per_thread), sizeof(int), 0, hipMemcpyHostToDevice));

    const int max_task_count = pixels_per_thread * thread_count;

    gpuErrchk(hipMalloc((void**) &(instance->tasks_gpu), 32 * max_task_count));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_tasks), &(instance->tasks_gpu), sizeof(void*), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMalloc((void**) &(instance->trace_results_gpu), sizeof(TraceResult) * max_task_count));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_trace_results), &(instance->trace_results_gpu), sizeof(void*), 0, hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc((void**) &(instance->task_counts_gpu), 4 * sizeof(uint16_t) * thread_count));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_task_counts), &(instance->task_counts_gpu), sizeof(void*), 0, hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc((void**) &(instance->randoms_gpu), sizeof(hiprandStateXORWOW_t) * thread_count));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_sample_randoms), &(instance->randoms_gpu), sizeof(void*), 0, hipMemcpyHostToDevice));

    initialize_randoms<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>();

    return instance;
}

extern "C" void* initialize_textures(TextureRGBA* textures, const int textures_length) {
    hipTextureObject_t* textures_cpu = (hipTextureObject_t*) malloc(sizeof(hipTextureObject_t) * textures_length);
    hipTextureObject_t* textures_gpu;

    gpuErrchk(hipMalloc((void**) &(textures_gpu), sizeof(hipTextureObject_t) * textures_length));

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0]   = hipAddressModeWrap;
    texDesc.addressMode[1]   = hipAddressModeWrap;
    texDesc.filterMode       = hipFilterModeLinear;
    texDesc.readMode         = hipReadModeElementType;
    texDesc.normalizedCoords = 1;

    for (int i = 0; i < textures_length; i++) {
        TextureRGBA texture = textures[i];

        const int num_rows = texture.height;
        const int num_cols = texture.width;
        RGBAF* data = texture.data;
        RGBAF* data_gpu;
        size_t pitch;
        gpuErrchk(hipMallocPitch((void**) &data_gpu, &pitch, num_cols * sizeof(RGBAF), num_rows));
        gpuErrchk(hipMemcpy2D(data_gpu, pitch, data, num_cols * sizeof(RGBAF), num_cols * sizeof(RGBAF), num_rows, hipMemcpyHostToDevice));

        struct hipResourceDesc resDesc;
        memset(&resDesc, 0, sizeof(resDesc));
        resDesc.resType = hipResourceTypePitch2D;
        resDesc.res.pitch2D.devPtr = data_gpu;
        resDesc.res.pitch2D.width = num_cols;
        resDesc.res.pitch2D.height = num_rows;
        resDesc.res.pitch2D.desc = hipCreateChannelDesc<float4>();
        resDesc.res.pitch2D.pitchInBytes = pitch;

        gpuErrchk(hipCreateTextureObject(textures_cpu + i, &resDesc, &texDesc, NULL));
    }

    gpuErrchk(hipMemcpy(textures_gpu, textures_cpu, sizeof(hipTextureObject_t) * textures_length, hipMemcpyHostToDevice));

    free(textures_cpu);

    return textures_gpu;
}

extern "C" void initialize_device() {
    gpuErrchk(hipSetDeviceFlags(hipDeviceMapHost));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    printf("Luminary - %s\n",prop.name);
}

extern "C" void free_textures(void* texture_atlas, const int textures_length) {
    hipTextureObject_t* textures_cpu = (hipTextureObject_t*) malloc(sizeof(hipTextureObject_t) * textures_length);
    gpuErrchk(hipMemcpy(textures_cpu, texture_atlas, sizeof(hipTextureObject_t) * textures_length, hipMemcpyDeviceToHost));

    for (int i = 0; i < textures_length; i++) {
        gpuErrchk(hipDestroyTextureObject(textures_cpu[i]));
    }

    gpuErrchk(hipFree(texture_atlas));
    free(textures_cpu);
}

extern "C" void copy_framebuffer_to_cpu(RaytraceInstance* instance) {
    gpuErrchk(hipMemcpy(instance->frame_output, instance->frame_output_gpu, sizeof(RGBF) * instance->width * instance->height, hipMemcpyDeviceToHost));
}

extern "C" void trace_scene(RaytraceInstance* instance, const int temporal_frames, const unsigned int update_mask) {
    const int amount = instance->width * instance->height;

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_temporal_frames), &(temporal_frames), sizeof(int), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_pixels_left), &(amount), sizeof(int), 0, hipMemcpyHostToDevice));

    if (update_mask & 0b1)
        gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_scene), &(instance->scene_gpu), sizeof(Scene), 0, hipMemcpyHostToDevice));
    if (update_mask & 0b10)
        gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_shading_mode), &(instance->shading_mode), sizeof(unsigned int), 0, hipMemcpyHostToDevice));
    if (update_mask & 0b100)
        update_sun(instance->scene_gpu);
    if (update_mask & 0b1000)
        update_camera_pos(instance->scene_gpu, instance->width, instance->height);
    if (update_mask & 0b10000)
        gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_default_material), &(instance->default_material), sizeof(RGBF), 0, hipMemcpyHostToDevice));

    clock_t t = clock();

    int pixels_left = amount;
    const float ratio = 1.0f/(amount);

    generate_trace_tasks<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>();

    while (pixels_left > 0) {
        if (pixels_left < amount)
            balance_trace_tasks<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>();
        preprocess_trace_tasks<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>();
        process_trace_tasks<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>();
        postprocess_trace_tasks<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>();
        process_geometry_tasks<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>();
        process_ocean_tasks<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>();
        process_sky_tasks<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>();

        gpuErrchk(hipMemcpyFromSymbol(&(pixels_left), device_pixels_left, sizeof(int), 0, hipMemcpyDeviceToHost));
        gpuErrchk(hipDeviceSynchronize());
    }

    finalize_samples<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>();
}

extern "C" void apply_bloom(RaytraceInstance* instance, RGBF* image) {
    if (instance->denoiser) {
        //bloom_kernel_split<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(image);
        bloom_kernel_blur_vertical<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(image);
        bloom_kernel_blur_horizontal<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(image);
    }
}

extern "C" void free_inputs(RaytraceInstance* instance) {
    gpuErrchk(hipFree(instance->scene_gpu.texture_assignments));
    gpuErrchk(hipFree(instance->scene_gpu.triangles));
    gpuErrchk(hipFree(instance->scene_gpu.traversal_triangles));
    gpuErrchk(hipFree(instance->scene_gpu.nodes));
    gpuErrchk(hipFree(instance->scene_gpu.lights));
    gpuErrchk(hipFree(instance->tasks_gpu));
    gpuErrchk(hipFree(instance->frame_buffer_gpu));
    gpuErrchk(hipFree(instance->frame_variance_gpu));
    gpuErrchk(hipFree(instance->frame_bias_cache_gpu));
    gpuErrchk(hipFree(instance->randoms_gpu));
}

extern "C" void free_outputs(RaytraceInstance* instance) {
    gpuErrchk(hipFree(instance->frame_output_gpu));
    _mm_free(instance->frame_output);

    if (instance->denoiser) {
        gpuErrchk(hipFree(instance->albedo_buffer_gpu));
        gpuErrchk(hipFree(instance->bloom_scratch_gpu));
    }

    free(instance);
}

extern "C" void initialize_8bit_frame(RaytraceInstance* instance, const int width, const int height) {
    gpuErrchk(hipMalloc((void**) &(instance->buffer_8bit_gpu), sizeof(RGB8) * width * height));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_frame_8bit), &(instance->buffer_8bit_gpu), sizeof(RGB8*), 0, hipMemcpyHostToDevice));
}

extern "C" void free_8bit_frame(RaytraceInstance* instance) {
    gpuErrchk(hipFree(instance->buffer_8bit_gpu));
}

extern "C" void copy_framebuffer_to_8bit(RGB8* buffer, const int width, const int height, RGBF* source, RaytraceInstance* instance) {
    convert_RGBF_to_RGB8<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(width, height, source);
    gpuErrchk(hipMemcpy(buffer, instance->buffer_8bit_gpu, sizeof(RGB8) * width * height, hipMemcpyDeviceToHost));
}
