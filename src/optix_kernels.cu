#include "hip/hip_runtime.h"
#define UTILS_NO_DEVICE_TABLE

#include "utils.h"

extern "C" static __constant__ DeviceConstantMemory device;

#include "math.cuh"
#include "memory.cuh"
#include "utils.cuh"

enum OptixAlphaResult {
  OPTIX_ALPHA_RESULT_OPAQUE      = 0,
  OPTIX_ALPHA_RESULT_SEMI        = 1,
  OPTIX_ALPHA_RESULT_TRANSPARENT = 2
} typedef OptixAlphaResult;

// Kernels must be named __[SEMANTIC]__..., for example, __raygen__...
// This can be found under function name prefix in the programming guide

extern "C" __global__ void __raygen__optix() {
  const uint3 idx  = optixGetLaunchIndex();
  const uint3 dimx = optixGetLaunchDimensions();

  const uint16_t trace_task_count = device.trace_count[idx.x + idx.y * dimx.x];

  unsigned int ray_flags;

  switch (device.iteration_type) {
    default:
    case TYPE_BOUNCE:
    case TYPE_CAMERA:
      ray_flags = OPTIX_RAY_FLAG_NONE;
      break;
    case TYPE_LIGHT:
      // Disable OMM opaque hits because we want to know if we hit something that is fully opaque so we can reject.
      ray_flags = OPTIX_RAY_FLAG_ENFORCE_ANYHIT;
      break;
  }

  for (int i = 0; i < trace_task_count; i++) {
    const int offset     = get_task_address2(idx.x, idx.y, i);
    const TraceTask task = load_trace_task(device.trace_tasks + offset);
    const float2 result  = __ldcs((float2*) (device.ptrs.trace_results + offset));
    const int pixel      = task.index.y * device.width + task.index.x;

    const float3 origin = make_float3(task.origin.x, task.origin.y, task.origin.z);
    const float3 ray    = make_float3(task.ray.x, task.ray.y, task.ray.z);

    const float tmax = result.x;

    if (device.iteration_type == TYPE_LIGHT) {
      unsigned int depth             = __float_as_uint(result.x);
      unsigned int hit_id            = __float_as_uint(result.y);
      unsigned int accumulated_alpha = __float_as_uint(1.0f);

      optixTrace(
        device.optix_bvh, origin, ray, 0.0f, tmax, 0.0f, OptixVisibilityMask(0xFFFF), ray_flags, 0, 0, 0, depth, hit_id, accumulated_alpha);

      RGBF record = load_RGBF(device.ptrs.light_records + pixel);
      record      = scale_color(record, __uint_as_float(accumulated_alpha));
      store_RGBF(device.ptrs.light_records + pixel, record);

      float2 trace_result = make_float2(__uint_as_float(depth), __uint_as_float(hit_id));
      __stcs((float2*) (device.ptrs.trace_results + offset), trace_result);
    }
    else {
      unsigned int depth  = __float_as_uint(result.x);
      unsigned int hit_id = __float_as_uint(result.y);
      unsigned int cost   = 0;

      optixTrace(device.optix_bvh, origin, ray, 0.0f, tmax, 0.0f, OptixVisibilityMask(0xFFFF), ray_flags, 0, 0, 0, depth, hit_id, cost);

      if (__uint_as_float(depth) < tmax) {
        float2 trace_result;

        if (device.shading_mode == SHADING_HEAT) {
          trace_result = make_float2(cost, __uint_as_float(hit_id));
        }
        else {
          trace_result = make_float2(__uint_as_float(depth), __uint_as_float(hit_id));
        }

        __stcs((float2*) (device.ptrs.trace_results + offset), trace_result);
      }
    }
  }
}

/*
 * Performs alpha test on triangle
 * @result 0 if opaque, 1 if transparent, 2 if alpha cutoff
 */
__device__ OptixAlphaResult optix_alpha_test(float& alpha) {
  const unsigned int hit_id = optixGetPrimitiveIndex();

  const uint32_t material_id = load_triangle_material_id(hit_id);
  const uint16_t tex         = __ldg(&(device.scene.materials[material_id].albedo_map));

  alpha = 1.0f;

  if (tex != TEXTURE_NONE) {
    const UV uv = load_triangle_tex_coords(hit_id, optixGetTriangleBarycentrics());

    alpha = tex2D<float4>(device.ptrs.albedo_atlas[tex].tex, uv.u, 1.0f - uv.v).w;

    if (alpha <= device.scene.material.alpha_cutoff) {
      return OPTIX_ALPHA_RESULT_TRANSPARENT;
    }

    if (alpha < 1.0f) {
      return OPTIX_ALPHA_RESULT_SEMI;
    }
  }

  return OPTIX_ALPHA_RESULT_OPAQUE;
}

extern "C" __global__ void __anyhit__optix() {
  if (device.iteration_type == TYPE_CAMERA) {
    optixSetPayload_2(optixGetPayload_2() + 1);
  }

  float alpha;
  const OptixAlphaResult alpha_result = optix_alpha_test(alpha);

  if (alpha_result == OPTIX_ALPHA_RESULT_TRANSPARENT) {
    optixIgnoreIntersection();
  }

  if (device.iteration_type == TYPE_LIGHT) {
    if (optixGetPrimitiveIndex() == optixGetPayload_1()) {
      optixIgnoreIntersection();
    }

    if (alpha_result == OPTIX_ALPHA_RESULT_OPAQUE) {
      optixSetPayload_0(__float_as_uint(0.0f));
      optixSetPayload_1(HIT_TYPE_REJECT);

      optixTerminateRay();
    }

    unsigned int accumulated_alpha = optixGetPayload_2();
    accumulated_alpha              = __float_as_uint((1.0f - alpha) * __uint_as_float(accumulated_alpha));
    optixSetPayload_2(accumulated_alpha);

    optixIgnoreIntersection();
  }
}

extern "C" __global__ void __closesthit__optix() {
  optixSetPayload_0(__float_as_uint(optixGetRayTmax()));
  optixSetPayload_1(optixGetPrimitiveIndex());
}
