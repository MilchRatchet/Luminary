#include "hip/hip_runtime.h"
#include "scene.h"
#include "primitives.h"
#include "image.h"
#include "raytrace.h"
#include "mesh.h"
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <float.h>
#include <stdio.h>

const int threads_per_block = 256;
const int blocks_per_grid = 960;

__device__
const float epsilon = 0.001f;

__device__
int device_reflection_depth;

__device__
Scene device_scene;

__device__
int device_diffuse_samples;

__device__
RGBF* device_frame;

__device__
unsigned int device_width;

__device__
unsigned int device_height;

__device__
unsigned int device_amount;

__device__
float device_step;

__device__
float device_vfov;

__device__
float device_offset_x;

__device__
float device_offset_y;

__device__
Quaternion device_camera_rotation;

__device__
hiprandStateXORWOW_t device_random;

__device__
vec3 cross_product(const vec3 a, const vec3 b) {
    vec3 result;

    result.x = a.y*b.z - a.z*b.y;
    result.y = a.z*b.x - a.x*b.z;
    result.z = a.x*b.y - a.y*b.x;

    return result;
}

__device__
float dot_product(const vec3 a, const vec3 b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__
vec3 vec_diff(const vec3 a, const vec3 b) {
    vec3 result;

    result.x = a.x - b.x;
    result.y = a.y - b.y;
    result.z = a.z - b.z;

    return result;
}

__device__
vec3 get_coordinates_in_triangle(const Triangle triangle, const vec3 point) {
    const vec3 e1 = vec_diff(triangle.v2, triangle.v1);
    const vec3 e2 = vec_diff(triangle.v3, triangle.v1);
    const vec3 diff = vec_diff(point, triangle.v1);
    const float d00 = dot_product(e1,e1);
    const float d01 = dot_product(e1,e2);
    const float d11 = dot_product(e2,e2);
    const float d20 = dot_product(diff,e1);
    const float d21 = dot_product(diff,e2);
    const float denom = 1.0f / (d00 * d11 - d01 * d01);
    vec3 result;
    result.x = (d11 * d20 - d01 * d21) * denom;
    result.y = (d00 * d21 - d01 * d20) * denom;
    return result;
}

/*
 * Normals should really be spherically lerped but in most cases
 * the angle between the vertex normals is small enough, so that
 * the performance advantage is probably worth it.
 */
__device__
vec3 lerp_normals(const Triangle triangle, const float lambda, const float mu) {
    const vec3 e1 = vec_diff(triangle.vn2, triangle.vn1);
    const vec3 e2 = vec_diff(triangle.vn3, triangle.vn1);

    vec3 result;

    result.x = triangle.vn1.x + lambda * e1.x + mu * e2.x;
    result.y = triangle.vn1.y + lambda * e1.y + mu * e2.y;
    result.z = triangle.vn1.z + lambda * e1.z + mu * e2.z;

    const float length = 1.0f / sqrtf(dot_product(result, result));

    result.x *= length;
    result.y *= length;
    result.z *= length;

    return result;
}

/*
 * Based on:
 * A. Majercik, C. Crassin, P. Shirley, M. McGuire,
 * "A Ray-Box Intersection Algorithm and Efficient Dynamic Voxel Rendering",
 * Journal of Computer Graphics Techniques, 7(3), pp. 66-82, 2018
 *
 * This implementation is probably not quite sufficient
 *
 * Assumes that origin is not inside a box
 */
/*__device__
float cuboid_intersect(Cuboid cuboid, vec3 origin, vec3 ray) {
    origin.x -= cuboid.pos.x;
    origin.y -= cuboid.pos.y;
    origin.z -= cuboid.pos.z;

    vec3 d;

    vec3 sign;

    sign.x = copysignf(1.0f, -ray.x);
    sign.y = copysignf(1.0f, -ray.y);
    sign.z = copysignf(1.0f, -ray.z);

    d.x = cuboid.size.x * sign.x - origin.x;
    d.y = cuboid.size.y * sign.y - origin.y;
    d.z = cuboid.size.z * sign.z - origin.z;

    d.x /= ray.x;
    d.y /= ray.y;
    d.z /= ray.z;

    const bool test_x = (d.x >= 0.0f) && (fabsf(origin.y + ray.y * d.x) < cuboid.size.y) && (fabsf(origin.z + ray.z * d.x) < cuboid.size.z);
    const bool test_y = (d.y >= 0.0f) && (fabsf(origin.x + ray.x * d.y) < cuboid.size.x) && (fabsf(origin.z + ray.z * d.y) < cuboid.size.z);
    const bool test_z = (d.z >= 0.0f) && (fabsf(origin.x + ray.x * d.z) < cuboid.size.x) && (fabsf(origin.y + ray.y * d.z) < cuboid.size.y);

    vec3 sgn;

    sgn.x = 0.0f;
    sgn.y = 0.0f;
    sgn.z = 0.0f;

    if (test_x) {
        sgn.x = sign.x;
    }
    else if (test_y) {
        sgn.y = sign.y;
    }
    else if (test_z) {
        sgn.z = sign.z;
    }

    if (sgn.x != 0.0f) {
        return d.x;
    }
    else if (sgn.y != 0.0f) {
        return d.y;
    }
    else if (sgn.z != 0.0f) {
        return d.z;
    }
    else {
        return FLT_MAX;
    }
}*/

/*
 * Based on
 *
 * Möller, Tomas, Trumbore, Ben,
 * "Fast, Minimum Storage Ray-Triangle Intersection",
 * Journal of Graphics Tools, 2, pp. 21-28, 1997.
 *
 * This suggests storing vertex1, edge1 and edge2 in triangle instead of the three vertices
 */
__device__
float triangle_intersection(Triangle triangle, vec3 origin, vec3 ray) {
    const vec3 e1 = vec_diff(triangle.v2, triangle.v1);
    const vec3 e2 = vec_diff(triangle.v3, triangle.v1);

    const vec3 h = cross_product(ray, e2);
    const float a = dot_product(e1, h);

    if (a > -epsilon && a < epsilon) return FLT_MAX;

    const float f = 1.0f / a;
    const vec3 s = vec_diff(origin, triangle.v1);
    const float u = f * dot_product(s, h);

    if (u < 0.0f || u > 1.0f) return FLT_MAX;

    const vec3 q = cross_product(s, e1);
    const float v = f * dot_product(ray, q);

    if (v < 0.0f || u + v > 1.0f) return FLT_MAX;

    const float t = f * dot_product(e2, q);

    if (t > epsilon) {
        return t;
    } else {
        return FLT_MAX;
    }
}

/*
 * Based on
 * J. Frisvad,
 * "Building an Orthonormal Basis from a 3D Unit Vector Without Normalization",
 * Journal of Graphics Tools, 16(3), pp. 151-159, 2012
 */
__device__
vec3 sample_ray_from_angles_and_vector(const float theta, const float phi, const vec3 basis) {
    vec3 u1, u2;
    if (basis.z < -0.9999999f) {
        u1.x = 0.0f;
        u1.y = -1.0f;
        u1.z = 0.0f;
        u2.x = -1.0f;
        u2.y = 0.0f;
        u2.z = 0.0f;
    }
    else
    {
        const float a = 1.0f/(1.0f+basis.z);
        const float b = -basis.x*basis.y*a;
        u1.x = 1.0f - basis.x*basis.x*a;
        u1.y = b;
        u1.z = -basis.x;
        u2.x = b;
        u2.y = 1.0f-basis.y*basis.y*a;
        u2.z = -basis.y;
    }


    const float c1 = sinf(theta) * cosf(phi);
    const float c2 = sinf(theta) * sinf(phi);
    const float c3 = cosf(theta);

    vec3 result;

    result.x = c1 * u1.x + c2 * u2.x + c3 * basis.x;
    result.y = c1 * u1.y + c2 * u2.y + c3 * basis.y;
    result.z = c1 * u1.z + c2 * u2.z + c3 * basis.z;

    return result;
}

/*
 * Uses some self modified Blinn-Phong BRDF.
 */
__device__
RGBF trace_ray_iterative(vec3 origin, vec3 ray, hiprandStateXORWOW_t* random) {
    RGBF result;
    result.r = 0.0f;
    result.g = 0.0f;
    result.b = 0.0f;

    float weight = 1.0f;
    RGBF record;
    record.r = 1.0f;
    record.g = 1.0f;
    record.b = 1.0f;

    RGBF sky;
    sky.r = 1.0f;
    sky.g = 1.0f;
    sky.b = 1.0f;

    for (int reflection_number = 0; reflection_number < device_reflection_depth; reflection_number++) {
        float depth = 1000.0f;

        unsigned int hit_id = 0xffffffff;

        vec3 curr;

        for (unsigned int i = 0; i < device_scene.triangles_length; i++) {
            const float d = triangle_intersection(device_scene.triangles[i], origin, ray);

            if (d < depth) {
                depth = d;
                hit_id = i;
            }
        }

        if (hit_id == 0xffffffff) {
            result.r += sky.r * weight * record.r;
            result.g += sky.g * weight * record.g;
            result.b += sky.b * weight * record.b;

            return result;
        }

        curr.x = origin.x + ray.x * depth;
        curr.y = origin.y + ray.y * depth;
        curr.z = origin.z + ray.z * depth;

        Triangle hit_triangle = device_scene.triangles[hit_id];

        RGBF albedo;
        albedo.r = 0.5f;
        albedo.g = 0.5f;
        albedo.b = 0.5f;
        vec3 normal;
        normal.x = 1.0f;
        normal.y = 1.0f;
        normal.z = 1.0f;
        const float smoothness = 0.0f;
        RGBF emission;
        emission.r = 0.0f;
        emission.g = 0.0f;
        emission.b = 0.0f;
        const float intensity = 0.0f;
        const float metallic = 1.0f;

        normal = get_coordinates_in_triangle(hit_triangle, curr);

        const float lambda = normal.x;
        const float mu = normal.y;

        normal = lerp_normals(hit_triangle, lambda, mu);

        result.r += emission.r * intensity * weight * record.r;
        result.g += emission.g * intensity * weight * record.g;
        result.b += emission.b * intensity * weight * record.b;

        curr.x += normal.x * epsilon * 2.0f;
        curr.y += normal.y * epsilon * 2.0f;
        curr.z += normal.z * epsilon * 2.0f;

        origin.x = curr.x;
        origin.y = curr.y;
        origin.z = curr.z;

        vec3 specular_ray;
        const float projected_length = ray.x * normal.x + ray.y * normal.y + ray.z * normal.z;

        specular_ray.x = ray.x - 2.0f * projected_length * normal.x;
        specular_ray.y = ray.y - 2.0f * projected_length * normal.y;
        specular_ray.z = ray.z - 2.0f * projected_length * normal.z;

        const float specular_ray_length = rsqrtf(specular_ray.x * specular_ray.x + specular_ray.y * specular_ray.y + specular_ray.z * specular_ray.z);

        specular_ray.x *= specular_ray_length;
        specular_ray.y *= specular_ray_length;
        specular_ray.z *= specular_ray_length;

        const float specular = hiprand_uniform(&device_random);

        if (specular < smoothness) {
            record.r *= (albedo.r * metallic + 1.0f - metallic);
            record.g *= (albedo.g * metallic + 1.0f - metallic);
            record.b *= (albedo.b * metallic + 1.0f - metallic);

            const float n = 100.0f * smoothness / (1.0f + epsilon - smoothness);

            const float alpha = acosf(__powf(hiprand_uniform(random),(1.0f/(1.0f+n))));
            const float gamma = 2.0f * 3.1415926535f * hiprand_uniform(random);

            ray = sample_ray_from_angles_and_vector(alpha, gamma, specular_ray);

            weight *= 2.0f * 3.1415926535f;
        }
        else
        {
            record.r *= albedo.r;
            record.g *= albedo.g;
            record.b *= albedo.b;

            const float alpha = acosf(__fsqrt_rn(hiprand_uniform(random)));
            const float gamma = 2.0f * 3.1415926535f * hiprand_uniform(random);

            ray = sample_ray_from_angles_and_vector(alpha, gamma, normal);

            const float angle = normal.x * ray.x + normal.y * ray.y + normal.z * ray.z;

            weight *= 3.1415926535f / (angle + epsilon);
        }

        const float angle = specular_ray.x * ray.x + specular_ray.y * ray.y + specular_ray.z * ray.z;

        weight *= ((1.0f - smoothness) * 0.31830988618f) + (smoothness * 0.5f * 0.31830988618f);
    }

    return result;
}

__device__
vec3 rotate_vector_by_quaternion(vec3 v, Quaternion q) {
    vec3 result;

    vec3 u;
    u.x = q.x;
    u.y = q.y;
    u.z = q.z;

    float s = q.w;

    float dot_uv = u.x * v.x + u.y * v.y + u.z * v.z;
    float dot_uu = u.x * u.x + u.y * u.y + u.z * u.z;

    vec3 cross;

    cross.x = u.y*v.z - u.z*v.y;
    cross.y = u.z*v.x - u.x*v.z;
    cross.z = u.x*v.y - u.y*v.x;

    result.x = 2.0f * dot_uv * u.x + ((s*s)-dot_uu) * v.x + 2.0f * s * cross.x;
    result.y = 2.0f * dot_uv * u.y + ((s*s)-dot_uu) * v.y + 2.0f * s * cross.y;
    result.z = 2.0f * dot_uv * u.z + ((s*s)-dot_uu) * v.z + 2.0f * s * cross.z;

    return result;
}

__global__
void trace_rays() {
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;

    while (id < device_amount) {
        int x = id % device_width;
        int y = (id - x) / device_width;

        hiprandStateXORWOW_t random;

        hiprand_init(id, 0, 0, &random);

        vec3 ray;
        vec3 default_ray;

        RGBF result;

        result.r = 0.0f;
        result.g = 0.0f;
        result.b = 0.0f;

        for (int i = 0; i < device_diffuse_samples; i++) {
            default_ray.x = -device_scene.camera.fov + device_step * x + device_offset_x * hiprand_uniform(&random) * 2.0f;
            default_ray.y = device_vfov - device_step * y - device_offset_y * hiprand_uniform(&random) * 2.0f;
            default_ray.z = -1.0f;

            ray = rotate_vector_by_quaternion(default_ray, device_camera_rotation);

            float ray_length = __frsqrt_rn(ray.x * ray.x + ray.y * ray.y + ray.z * ray.z);

            ray.x *= ray_length;
            ray.y *= ray_length;
            ray.z *= ray_length;

            RGBF color = trace_ray_iterative(device_scene.camera.pos, ray, &random);

            result.r += color.r;
            result.g += color.g;
            result.b += color.b;
        }

        float weight = 1.0f/(float)device_diffuse_samples;

        result.r *= weight;
        result.g *= weight;
        result.b *= weight;

        device_frame[id] = result;

        id += blockDim.x * gridDim.x;
    }
}

__global__
void set_up_raytracing_device() {
    device_amount = device_width * device_height;

    device_step = 2.0f * (device_scene.camera.fov / device_width);

    device_vfov = device_step * device_height / 2.0f;

    device_offset_x = (device_step / 2.0f);
    device_offset_y = (device_step / 2.0f);

    float alpha = device_scene.camera.rotation.x;
    float beta = device_scene.camera.rotation.y;
    float gamma = device_scene.camera.rotation.z;

    double cy = __cosf(gamma * 0.5);
    double sy = __sinf(gamma * 0.5);
    double cp = __cosf(beta * 0.5);
    double sp = __sinf(beta * 0.5);
    double cr = __cosf(alpha * 0.5);
    double sr = __sinf(alpha * 0.5);

    Quaternion q;
    q.w = cr * cp * cy + sr * sp * sy;
    q.x = sr * cp * cy - cr * sp * sy;
    q.y = cr * sp * cy + sr * cp * sy;
    q.z = cr * cp * sy - sr * sp * cy;

    device_camera_rotation = q;

    hiprand_init(0,0,0,&device_random);
}


extern "C" raytrace_instance* init_raytracing(const unsigned int width, const unsigned int height, const int reflection_depth, const int diffuse_samples) {
    raytrace_instance* instance = (raytrace_instance*)malloc(sizeof(raytrace_instance));

    instance->width = width;
    instance->height = height;
    instance->frame_buffer = (RGBF*)malloc(sizeof(RGBF) * width * height);

    hipMalloc((void**) &(instance->frame_buffer_gpu), sizeof(RGBF) * width * height);

    instance->reflection_depth = reflection_depth;
    instance->diffuse_samples = diffuse_samples;

    return instance;
}

extern "C" void trace_scene(Scene scene, raytrace_instance* instance) {
    Scene scene_gpu = scene;

    hipMalloc((void**) &(scene_gpu.triangles), sizeof(Triangle) * scene_gpu.triangles_length);
    puts(hipGetErrorString(hipGetLastError()));

    hipMemcpy(scene_gpu.triangles, scene.triangles, sizeof(Triangle) * scene.triangles_length, hipMemcpyHostToDevice);
    puts(hipGetErrorString(hipGetLastError()));

    hipMemcpyToSymbol(HIP_SYMBOL(device_frame), &(instance->frame_buffer_gpu), sizeof(RGBF*), 0, hipMemcpyHostToDevice);
    puts(hipGetErrorString(hipGetLastError()));
    hipMemcpyToSymbol(HIP_SYMBOL(device_scene), &scene_gpu, sizeof(Scene), 0, hipMemcpyHostToDevice);
    puts(hipGetErrorString(hipGetLastError()));
    hipMemcpyToSymbol(HIP_SYMBOL(device_reflection_depth), &(instance->reflection_depth), sizeof(int), 0, hipMemcpyHostToDevice);
    puts(hipGetErrorString(hipGetLastError()));
    hipMemcpyToSymbol(HIP_SYMBOL(device_diffuse_samples), &(instance->diffuse_samples), sizeof(int), 0, hipMemcpyHostToDevice);
    puts(hipGetErrorString(hipGetLastError()));
    hipMemcpyToSymbol(HIP_SYMBOL(device_width), &(instance->width), sizeof(unsigned int), 0, hipMemcpyHostToDevice);
    puts(hipGetErrorString(hipGetLastError()));
    hipMemcpyToSymbol(HIP_SYMBOL(device_height), &(instance->height), sizeof(unsigned int), 0, hipMemcpyHostToDevice);

    puts(hipGetErrorString(hipGetLastError()));

    set_up_raytracing_device<<<1,1>>>();

    hipDeviceSynchronize();

    puts(hipGetErrorString(hipGetLastError()));

    trace_rays<<<blocks_per_grid,threads_per_block>>>();

    hipDeviceSynchronize();

    puts(hipGetErrorString(hipGetLastError()));

    hipMemcpy(instance->frame_buffer, instance->frame_buffer_gpu, sizeof(RGBF) * instance->width * instance->height, hipMemcpyDeviceToHost);

    puts(hipGetErrorString(hipGetLastError()));

    hipFree(scene_gpu.triangles);
}

extern "C" void frame_buffer_to_image(Camera camera, raytrace_instance* instance, RGB8* image) {
    for (int j = 0; j < instance->height; j++) {
        for (int i = 0; i < instance->width; i++) {
            RGB8 pixel;
            RGBF pixel_float = instance->frame_buffer[i + instance->width * j];

            pixel.r = (uint8_t)(min(255.9f, pixel_float.r * 255.9f));
            pixel.g = (uint8_t)(min(255.9f, pixel_float.g * 255.9f));
            pixel.b = (uint8_t)(min(255.9f, pixel_float.b * 255.9f));


            image[i + instance->width * j] = pixel;
        }
    }
}

extern "C" void free_raytracing(raytrace_instance* instance) {
    hipFree(instance->frame_buffer_gpu);

    free(instance->frame_buffer);

    free(instance);
}
