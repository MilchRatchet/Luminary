#include "hip/hip_runtime.h"
#include "math.cuh"
#include "memory.cuh"
#include "ocean.cuh"
#include "utils.cuh"

__global__ __launch_bounds__(THREADS_PER_BLOCK, 10) void process_debug_ocean_tasks() {
  const int task_count  = device.ptrs.task_counts[THREAD_ID * TASK_ADDRESS_COUNT_STRIDE + TASK_ADDRESS_OFFSET_OCEAN];
  const int task_offset = device.ptrs.task_offsets[THREAD_ID * TASK_ADDRESS_OFFSET_STRIDE + TASK_ADDRESS_OFFSET_OCEAN];

  for (int i = 0; i < task_count; i++) {
    OceanTask task  = load_ocean_task(device.trace_tasks + get_task_address(task_offset + i));
    const int pixel = task.index.y * device.width + task.index.x;

    if (device.shading_mode == SHADING_DEPTH) {
      const float dist  = get_length(sub_vector(device.scene.camera.pos, task.position));
      const float value = __saturatef((1.0f / dist) * 2.0f);
      write_beauty_buffer(get_color(value, value, value), pixel, true);
    }
    else if (device.shading_mode == SHADING_NORMAL) {
      vec3 normal = ocean_get_normal(task.position);

      normal.x = 0.5f * normal.x + 0.5f;
      normal.y = 0.5f * normal.y + 0.5f;
      normal.z = 0.5f * normal.z + 0.5f;

      write_beauty_buffer(get_color(__saturatef(normal.x), __saturatef(normal.y), __saturatef(normal.z)), pixel, true);
    }
    else if (device.shading_mode == SHADING_IDENTIFICATION) {
      write_beauty_buffer(get_color(0.0f, 0.0f, 1.0f), pixel, true);
    }
  }
}
