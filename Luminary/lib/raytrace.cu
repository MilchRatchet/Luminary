#include "hip/hip_runtime.h"
#include "scene.h"
#include "primitives.h"
#include "image.h"
#include "raytrace.h"
#include "mesh.h"
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <float.h>
#include <stdio.h>
#include <time.h>
#include <chrono>
#include <thread>
#include <immintrin.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

const int threads_per_block = 128;
const int blocks_per_grid = 512;

__device__
const float epsilon = 0.001f;

__device__
int device_reflection_depth;

__device__
Scene device_scene;

__device__
int device_diffuse_samples;

__device__
RGBF* device_frame;

__device__
unsigned int device_width;

__device__
unsigned int device_height;

__device__
unsigned int device_amount;

__device__
float device_step;

__device__
float device_vfov;

__device__
float device_offset_x;

__device__
float device_offset_y;

__device__
Quaternion device_camera_rotation;

__device__
hiprandStateXORWOW_t device_random;

__device__
hipTextureObject_t* device_albedo_atlas;

__device__
hipTextureObject_t* device_illuminance_atlas;

__device__
hipTextureObject_t* device_material_atlas;

__device__
texture_assignment* device_texture_assignments;

__device__
vec3 cross_product(const vec3 a, const vec3 b) {
    vec3 result;

    result.x = a.y*b.z - a.z*b.y;
    result.y = a.z*b.x - a.x*b.z;
    result.z = a.x*b.y - a.y*b.x;

    return result;
}

__device__
float dot_product(const vec3 a, const vec3 b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__
vec3 vec_diff(const vec3 a, const vec3 b) {
    vec3 result;

    result.x = a.x - b.x;
    result.y = a.y - b.y;
    result.z = a.z - b.z;

    return result;
}

__device__
vec3 get_coordinates_in_triangle(const Triangle triangle, const vec3 point) {
    const vec3 diff = vec_diff(point, triangle.vertex);
    const float d00 = dot_product(triangle.edge1,triangle.edge1);
    const float d01 = dot_product(triangle.edge1,triangle.edge2);
    const float d11 = dot_product(triangle.edge2,triangle.edge2);
    const float d20 = dot_product(diff,triangle.edge1);
    const float d21 = dot_product(diff,triangle.edge2);
    const float denom = 1.0f / (d00 * d11 - d01 * d01);
    vec3 result;
    result.x = (d11 * d20 - d01 * d21) * denom;
    result.y = (d00 * d21 - d01 * d20) * denom;
    return result;
}

/*
 * Normals should really be spherically lerped but in most cases
 * the angle between the vertex normals is small enough, so that
 * the performance advantage is probably worth it.
 */
__device__
vec3 lerp_normals(const Triangle triangle, const float lambda, const float mu) {
    vec3 result;

    result.x = triangle.vertex_normal.x + lambda * triangle.edge1_normal.x + mu * triangle.edge2_normal.x;
    result.y = triangle.vertex_normal.y + lambda * triangle.edge1_normal.y + mu * triangle.edge2_normal.y;
    result.z = triangle.vertex_normal.z + lambda * triangle.edge1_normal.z + mu * triangle.edge2_normal.z;

    const float length = 1.0f / sqrtf(dot_product(result, result));

    result.x *= length;
    result.y *= length;
    result.z *= length;

    return result;
}

__device__
UV lerp_uv(const Triangle triangle, const float lambda, const float mu) {
    UV result;

    result.u = triangle.vertex_texture.u + lambda * triangle.edge1_texture.u + mu * triangle.edge2_texture.u;
    result.v = triangle.vertex_texture.v + lambda * triangle.edge1_texture.v + mu * triangle.edge2_texture.v;

    return result;
}

__device__
float ray_box_intersect(const vec3 low, const vec3 high, vec3 origin, const vec3 ray) {
    origin.x -= (high.x + low.x) * 0.5f;
    origin.y -= (high.y + low.y) * 0.5f;
    origin.z -= (high.z + low.z) * 0.5f;

    const float size_x = (high.x - low.x) * 0.5f;
    const float size_y = (high.y - low.y) * 0.5f;
    const float size_z = (high.z - low.z) * 0.5f;

    if ((fabsf(origin.x) < size_x) && (fabsf(origin.y) < size_y) && (fabsf(origin.z) < size_z)) return 0.0f;

    vec3 d;

    vec3 sign;

    sign.x = copysignf(1.0f, -ray.x);
    sign.y = copysignf(1.0f, -ray.y);
    sign.z = copysignf(1.0f, -ray.z);

    d.x = size_x * sign.x - origin.x;
    d.y = size_y * sign.y - origin.y;
    d.z = size_z * sign.z - origin.z;

    d.x /= ray.x;
    d.y /= ray.y;
    d.z /= ray.z;

    const bool test_x = (d.x >= 0.0f) && (fabsf(origin.y + ray.y * d.x) < size_y) && (fabsf(origin.z + ray.z * d.x) < size_z);
    const bool test_y = (d.y >= 0.0f) && (fabsf(origin.x + ray.x * d.y) < size_x) && (fabsf(origin.z + ray.z * d.y) < size_z);
    const bool test_z = (d.z >= 0.0f) && (fabsf(origin.x + ray.x * d.z) < size_x) && (fabsf(origin.y + ray.y * d.z) < size_y);

    vec3 sgn;

    sgn.x = 0.0f;
    sgn.y = 0.0f;
    sgn.z = 0.0f;

    if (test_x) {
        sgn.x = sign.x;
    }
    else if (test_y) {
        sgn.y = sign.y;
    }
    else if (test_z) {
        sgn.z = sign.z;
    }

    if (sgn.x != 0.0f) {
        return d.x;
    }
    else if (sgn.y != 0.0f) {
        return d.y;
    }
    else if (sgn.z != 0.0f) {
        return d.z;
    }
    else {
        return FLT_MAX;
    }
}

__device__
float triangle_intersection(const Triangle triangle, const vec3 origin, const vec3 ray) {
    const vec3 h = cross_product(ray, triangle.edge2);
    const float a = dot_product(triangle.edge1, h);

    if (a > -0.00000001 && a < 0.00000001) return FLT_MAX;

    const float f = 1.0f / a;
    const vec3 s = vec_diff(origin, triangle.vertex);
    const float u = f * dot_product(s, h);

    if (u < 0.0f || u > 1.0f) return FLT_MAX;

    const vec3 q = cross_product(s, triangle.edge1);
    const float v = f * dot_product(ray, q);

    if (v < 0.0f || u + v > 1.0f) return FLT_MAX;

    const float t = f * dot_product(triangle.edge2, q);

    if (t > -epsilon) {
        return t;
    } else {
        return FLT_MAX;
    }
}

__device__
vec3 sample_ray_from_angles_and_vector(const float theta, const float phi, const vec3 basis) {
    vec3 u1, u2;
    if (basis.z < -0.9999999f) {
        u1.x = 0.0f;
        u1.y = -1.0f;
        u1.z = 0.0f;
        u2.x = -1.0f;
        u2.y = 0.0f;
        u2.z = 0.0f;
    }
    else
    {
        const float a = 1.0f/(1.0f+basis.z);
        const float b = -basis.x*basis.y*a;
        u1.x = 1.0f - basis.x*basis.x*a;
        u1.y = b;
        u1.z = -basis.x;
        u2.x = b;
        u2.y = 1.0f-basis.y*basis.y*a;
        u2.z = -basis.y;
    }


    const float c1 = sinf(theta) * cosf(phi);
    const float c2 = sinf(theta) * sinf(phi);
    const float c3 = cosf(theta);

    vec3 result;

    result.x = c1 * u1.x + c2 * u2.x + c3 * basis.x;
    result.y = c1 * u1.y + c2 * u2.y + c3 * basis.y;
    result.z = c1 * u1.z + c2 * u2.z + c3 * basis.z;

    return result;
}

__device__
int trailing_zeros(const unsigned int n) {
    int mask = 1;
    for (int i = 0; i < 32; i++, mask <<= 1)
        if ((n & mask) != 0)
            return i;

    return 32;
}

__device__
vec3 decompress_vector(const compressed_vec3 vector, const vec3 p, const float ex, const float ey, const float ez) {
    vec3 result;

    result.x = p.x + ex * (float)vector.x;
    result.y = p.y + ey * (float)vector.y;
    result.z = p.z + ez * (float)vector.z;

    return result;
}


__device__
vec3 scale_vector(vec3 vector, const float scale) {
    vector.x *= scale;
    vector.y *= scale;
    vector.z *= scale;

    return vector;
}

__device__
float get_length(const vec3 vector) {
    return sqrtf(vector.x * vector.x + vector.y * vector.y + vector.z * vector.z);
}

__device__
vec3 normalize_vector(vec3 vector) {
    const float inv_length = 1.0f / get_length(vector);

    return scale_vector(vector, inv_length);
}

__device__
float get_length_to_border(const vec3 origin, const vec3 ray, const float atmosphere_end) {
    const float a = dot_product(origin,ray);
    return -a + sqrtf(a * a - dot_product(origin,origin) + atmosphere_end * atmosphere_end);
}

__device__
float density_at_height(const float height, const float density_falloff) {
    return expf(-height * density_falloff);
}

__device__
float height_at_point(const vec3 point) {
    const float earth_radius = 6371.0f;
    const float atmosphere_height = 100.0f;
    return (get_length(point) - earth_radius);
}


__device__
float get_optical_depth(const vec3 origin, const vec3 ray, const float length) {
    if (length == 0.0f) return 0.0f;

    const int steps = 8;
    const float step_size = length / steps;
    float depth = 0.0f;
    vec3 point = origin;

    point.x += step_size * ray.x * 0.125f;
    point.y += step_size * ray.y * 0.125f;
    point.z += step_size * ray.z * 0.125f;

    for (int i = 0; i < steps; i++) {
        depth += density_at_height(height_at_point(point),0.125f) * step_size;

        point.x += step_size * ray.x;
        point.y += step_size * ray.y;
        point.z += step_size * ray.z;
    }

    return depth;
}

__device__
RGBF get_sky_color(const vec3 ray) {
    RGBF result;
    result.r = 0.0f;
    result.g = 0.0f;
    result.b = 0.0f;

    const float angular_diameter = 0.018f;

    const float overall_density = 1.0f;

    RGBF scatter;
    scatter.r = 5.8f * 0.001f * overall_density;
    scatter.g = 13.558f * 0.001f * overall_density;
    scatter.b = 33.1f * 0.001f * overall_density;

    const float mie_scatter = 3.996f * 0.001f * overall_density;

    RGBF ozone_absorbtion;
    ozone_absorbtion.r = 0.65f * 0.001f * overall_density;
    ozone_absorbtion.g = 1.881f * 0.001f * overall_density;
    ozone_absorbtion.b = 0.085f * 0.001f * overall_density;

    const float sun_dist = 150000000.0f;

    RGBF sun_color;

    const float sun_intensity = 6.0f;

    sun_color.r = 1.0f * sun_intensity;
    sun_color.g = 0.9f * sun_intensity;
    sun_color.b = 0.8f * sun_intensity;

    vec3 sun;
    sun.x = 0.2f;
    sun.y = 0.05f;
    sun.z = -1.0f;

    const vec3 sun_normalized = normalize_vector(sun);
    sun = scale_vector(sun_normalized, sun_dist);

    const float earth_radius = 6371.0f;
    const float atmosphere_height = 100.0f;

    vec3 origin;
    origin.x = 0.0f;
    origin.y = earth_radius + 0.0f;
    origin.z = 0.0f;

    const vec3 origin_default = origin;

    const float limit = get_length_to_border(origin, ray, earth_radius + atmosphere_height);
    const int steps = 10;
    const float step_size = limit/steps;
    float reach = 0.0f;

    reach += step_size * 0.125f;

    origin.x += step_size * ray.x * 0.125f;
    origin.y += step_size * ray.y * 0.125f;
    origin.z += step_size * ray.z * 0.125f;

    for (int i = 0; i < steps; i++) {
        const vec3 ray_scatter = normalize_vector(vec_diff(sun, origin));

        const float optical_depth = get_optical_depth(origin_default, ray, reach) + get_optical_depth(origin, ray_scatter, get_length_to_border(origin, ray_scatter, earth_radius + atmosphere_height));

        const float height = height_at_point(origin);

        const float local_density = density_at_height(height, 0.125f);
        const float mie_density = density_at_height(height, 0.83333f);
        //The tent function is disabled atm, first argument 0.0f to activate
        const float ozone_density = fmaxf(1.0f, 1.0f - fabsf(height - 25.0f) * 0.066666667f);

        RGBF transmittance;
        transmittance.r = expf(-optical_depth * (scatter.r + ozone_density * ozone_absorbtion.r + 1.11f * mie_scatter));
        transmittance.g = expf(-optical_depth * (scatter.g + ozone_density * ozone_absorbtion.g + 1.11f * mie_scatter));
        transmittance.b = expf(-optical_depth * (scatter.b + ozone_density * ozone_absorbtion.b + 1.11f * mie_scatter));

        float cos_angle = dot_product(ray, ray_scatter);

        cos_angle = cosf(fmaxf(0.0f,acosf(cos_angle) - angular_diameter));

        const float rayleigh = 3.0f * (1.0f + cos_angle * cos_angle) / (16.0f * 3.1415926535f);

        const float g = 0.8f;
        const float mie = 1.5f * (1.0f + cos_angle * cos_angle) * (1.0f - g * g) / (4.0f * 3.1415926535f * (2.0f + g * g) * powf(1.0f + g * g - 2.0f * g * cos_angle, 1.5f));

        result.r += sun_color.r * transmittance.r * (local_density * scatter.r * rayleigh + mie_density * mie_scatter * mie) * step_size;
        result.g += sun_color.g * transmittance.g * (local_density * scatter.g * rayleigh + mie_density * mie_scatter * mie) * step_size;
        result.b += sun_color.b * transmittance.b * (local_density * scatter.b * rayleigh + mie_density * mie_scatter * mie) * step_size;

        reach += step_size;

        origin.x += step_size * ray.x;
        origin.y += step_size * ray.y;
        origin.z += step_size * ray.z;
    }
    const vec3 ray_sun = normalize_vector(vec_diff(sun, origin_default));

    float cos_angle = dot_product(ray, ray_sun);
    cos_angle = cosf(fmaxf(0.0f,acosf(cos_angle) - angular_diameter));

    if (cos_angle >= 0.99999f) {
        const float optical_depth = get_optical_depth(origin_default, ray, limit);

        const float height = height_at_point(origin_default);

        const float ozone_density = fmaxf(0.0f, 1.0f - fabsf(height - 25.0f) * 0.066666667f);

        RGBF transmittance;
        transmittance.r = expf(-optical_depth * (scatter.r + ozone_density * ozone_absorbtion.r + 1.11f * mie_scatter));
        transmittance.g = expf(-optical_depth * (scatter.g + ozone_density * ozone_absorbtion.g + 1.11f * mie_scatter));
        transmittance.b = expf(-optical_depth * (scatter.b + ozone_density * ozone_absorbtion.b + 1.11f * mie_scatter));

        result.r += sun_color.r * transmittance.r * cos_angle * 50.0f;
        result.g += sun_color.g * transmittance.g * cos_angle * 50.0f;
        result.b += sun_color.b * transmittance.b * cos_angle * 50.0f;
    }

    return result;
}


__device__
RGBF trace_ray_iterative(vec3 origin, vec3 ray, hiprandStateXORWOW_t* random) {
    RGBF result;
    result.r = 0.0f;
    result.g = 0.0f;
    result.b = 0.0f;

    float weight = 1.0f;
    RGBF record;
    record.r = 1.0f;
    record.g = 1.0f;
    record.b = 1.0f;

    int traversals = 0;
    unsigned int ray_triangle_intersections = 0;

    for (int reflection_number = 0; reflection_number < device_reflection_depth; reflection_number++) {
        float depth = device_scene.far_clip_distance;

        unsigned int hit_id = 0xffffffff;

        vec3 curr;

        int node_address = 0;
        int node_key = 1;
        int bit_trail = 0;
        int mrpn_address = -1;

        while (node_address != -1) {
            while (device_scene.nodes[node_address].triangles_address == -1) {
                Node node = device_scene.nodes[node_address];

                traversals++;

                const float decompression_x = __powf(2.0f, (float)node.ex);
                const float decompression_y = __powf(2.0f, (float)node.ey);
                const float decompression_z = __powf(2.0f, (float)node.ez);

                const vec3 left_high = decompress_vector(node.left_high, node.p, decompression_x, decompression_y, decompression_z);
                const vec3 left_low = decompress_vector(node.left_low, node.p, decompression_x, decompression_y, decompression_z);
                const vec3 right_high = decompress_vector(node.right_high, node.p, decompression_x, decompression_y, decompression_z);
                const vec3 right_low = decompress_vector(node.right_low, node.p, decompression_x, decompression_y, decompression_z);

                const float L = ray_box_intersect(left_low, left_high, origin, ray);
                const float R = ray_box_intersect(right_low, right_high, origin, ray);
                const int R_is_closest = R < L;

                if (L < depth || R < depth) {

                    node_key = node_key << 1;
                    bit_trail = bit_trail << 1;

                    if (L >= depth || R_is_closest) {
                        node_address = 2 * node_address + 2;
                        node_key = node_key ^ 0b1;
                    }
                    else {
                        node_address = 2 * node_address + 1;
                    }

                    if (L < depth && R < depth) {
                        bit_trail = bit_trail ^ 0b1;
                        if (R_is_closest) {
                            mrpn_address = node_address - 1;
                        }
                        else {
                            mrpn_address = node_address + 1;
                        }
                    }
                } else {
                    break;
                }
            }

            if (device_scene.nodes[node_address].triangles_address != -1) {
                Node node = device_scene.nodes[node_address];

                for (unsigned int i = 0; i < node.triangle_count; i++) {
                    const float d = triangle_intersection(device_scene.triangles[node.triangles_address + i], origin, ray);

                    ray_triangle_intersections++;

                    if (d < depth) {
                        depth = d;
                        hit_id = node.triangles_address + i;
                    }
                }
            }

            if (bit_trail == 0) {
                node_address = -1;
            }
            else {
                int num_levels = trailing_zeros(bit_trail);
                bit_trail = (bit_trail >> num_levels) ^ 0b1;
                node_key = (node_key >> num_levels) ^ 0b1;
                if (mrpn_address != -1) {
                    node_address = mrpn_address;
                    mrpn_address = -1;
                }
                else {
                    node_address = node_key - 1;
                }
            }
        }

        if (hit_id == 0xffffffff) {
            RGBF sky = get_sky_color(ray);

            result.r += sky.r * weight * record.r;
            result.g += sky.g * weight * record.g;
            result.b += sky.b * weight * record.b;

            /*result.r = ray_triangle_intersections * (16.0f/device_scene.triangles_length) * (1.0f/device_reflection_depth);
            result.g = 0;
            result.b = traversals * (4.0f/device_scene.nodes_length);*/

            return result;
        } /*else {
            result.r += depth * 0.05f;
            result.g += depth * 0.05f;
            result.b += depth * 0.05f;

            return result;
        }*/

        curr.x = origin.x + ray.x * depth;
        curr.y = origin.y + ray.y * depth;
        curr.z = origin.z + ray.z * depth;

        Triangle hit_triangle = device_scene.triangles[hit_id];

        vec3 normal;
        normal.x = 1.0f;
        normal.y = 1.0f;
        normal.z = 1.0f;


        normal = get_coordinates_in_triangle(hit_triangle, curr);

        const float lambda = normal.x;
        const float mu = normal.y;

        normal = lerp_normals(hit_triangle, lambda, mu);

        UV tex_coords = lerp_uv(hit_triangle, lambda, mu);

        RGBAF albedo;
        RGBF emission;

        float4 albedo_f = tex2D<float4>(device_albedo_atlas[device_texture_assignments[hit_triangle.object_maps].albedo_map], tex_coords.u, 1.0f - tex_coords.v);
        float4 illumiance_f = tex2D<float4>(device_illuminance_atlas[device_texture_assignments[hit_triangle.object_maps].illuminance_map], tex_coords.u, 1.0f - tex_coords.v);
        float4 material_f = tex2D<float4>(device_material_atlas[device_texture_assignments[hit_triangle.object_maps].material_map], tex_coords.u, 1.0f - tex_coords.v);

        albedo.r = albedo_f.x;
        albedo.g = albedo_f.y;
        albedo.b = albedo_f.z;
        albedo.a = albedo_f.w;

        emission.r = illumiance_f.x;
        emission.g = illumiance_f.y;
        emission.b = illumiance_f.z;

        const float smoothness = material_f.x;
        const float metallic = material_f.y;
        const float intensity = material_f.z;

        result.r += emission.r * intensity * weight * record.r;
        result.g += emission.g * intensity * weight * record.g;
        result.b += emission.b * intensity * weight * record.b;

        const float transparent_pass = hiprand_uniform(random);

        if (transparent_pass > albedo.a) {
            origin.x = curr.x + 2.0f * epsilon * ray.x;
            origin.y = curr.y + 2.0f * epsilon * ray.y;
            origin.z = curr.z + 2.0f * epsilon * ray.z;

            continue;
        }

        curr.x += normal.x * epsilon * 2.0f;
        curr.y += normal.y * epsilon * 2.0f;
        curr.z += normal.z * epsilon * 2.0f;

        origin.x = curr.x;
        origin.y = curr.y;
        origin.z = curr.z;

        vec3 specular_ray;
        const float projected_length = ray.x * normal.x + ray.y * normal.y + ray.z * normal.z;

        specular_ray.x = ray.x - 2.0f * projected_length * normal.x;
        specular_ray.y = ray.y - 2.0f * projected_length * normal.y;
        specular_ray.z = ray.z - 2.0f * projected_length * normal.z;

        const float specular_ray_length = rsqrtf(specular_ray.x * specular_ray.x + specular_ray.y * specular_ray.y + specular_ray.z * specular_ray.z);

        specular_ray.x *= specular_ray_length;
        specular_ray.y *= specular_ray_length;
        specular_ray.z *= specular_ray_length;

        const float specular = hiprand_uniform(random);

        if (specular < smoothness) {
            record.r *= (albedo.r * metallic + 1.0f - metallic);
            record.g *= (albedo.g * metallic + 1.0f - metallic);
            record.b *= (albedo.b * metallic + 1.0f - metallic);

            const float n = 100.0f * smoothness / (1.0f + epsilon - smoothness);

            const float alpha = acosf(__powf(hiprand_uniform(random),(1.0f/(1.0f+n))));
            const float gamma = 2.0f * 3.1415926535f * hiprand_uniform(random);

            ray = sample_ray_from_angles_and_vector(alpha, gamma, specular_ray);

            weight *= 2.0f * 3.1415926535f;
        }
        else
        {
            record.r *= albedo.r;
            record.g *= albedo.g;
            record.b *= albedo.b;

            const float alpha = acosf(__fsqrt_rn(hiprand_uniform(random)));
            const float gamma = 2.0f * 3.1415926535f * hiprand_uniform(random);

            ray = sample_ray_from_angles_and_vector(alpha, gamma, normal);

            const float angle = normal.x * ray.x + normal.y * ray.y + normal.z * ray.z;

            weight *= 3.1415926535f / (angle + epsilon);
        }

        const float angle = specular_ray.x * ray.x + specular_ray.y * ray.y + specular_ray.z * ray.z;

        weight *= ((1.0f - smoothness) * 0.31830988618f) + (smoothness * 0.5f * 0.31830988618f);
    }

    /*result.r = ray_triangle_intersections * (16.0f/device_scene.triangles_length) * (1.0f/device_reflection_depth);
    result.g = 0;
    result.b = traversals * (4.0f/device_scene.nodes_length);*/

    return result;
}

__device__
vec3 rotate_vector_by_quaternion(vec3 v, Quaternion q) {
    vec3 result;

    vec3 u;
    u.x = q.x;
    u.y = q.y;
    u.z = q.z;

    float s = q.w;

    float dot_uv = u.x * v.x + u.y * v.y + u.z * v.z;
    float dot_uu = u.x * u.x + u.y * u.y + u.z * u.z;

    vec3 cross;

    cross.x = u.y*v.z - u.z*v.y;
    cross.y = u.z*v.x - u.x*v.z;
    cross.z = u.x*v.y - u.y*v.x;

    result.x = 2.0f * dot_uv * u.x + ((s*s)-dot_uu) * v.x + 2.0f * s * cross.x;
    result.y = 2.0f * dot_uv * u.y + ((s*s)-dot_uu) * v.y + 2.0f * s * cross.y;
    result.z = 2.0f * dot_uv * u.z + ((s*s)-dot_uu) * v.z + 2.0f * s * cross.z;

    return result;
}

__global__
void trace_rays(volatile uint32_t* progress) {
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;

    while (id < device_amount) {
        int x = id % device_width;
        int y = id / device_width;

        hiprandStateXORWOW_t random;

        hiprand_init(id, 0, 0, &random);

        vec3 ray;
        vec3 default_ray;

        RGBF result;

        result.r = 0.0f;
        result.g = 0.0f;
        result.b = 0.0f;

        for (int i = 0; i < device_diffuse_samples; i++) {
            default_ray.x = -device_scene.camera.fov + device_step * x + device_offset_x * hiprand_uniform(&random) * 2.0f;
            default_ray.y = device_vfov - device_step * y - device_offset_y * hiprand_uniform(&random) * 2.0f;
            default_ray.z = -1.0f;

            ray = rotate_vector_by_quaternion(default_ray, device_camera_rotation);

            float ray_length = __frsqrt_rn(ray.x * ray.x + ray.y * ray.y + ray.z * ray.z);

            ray.x *= ray_length;
            ray.y *= ray_length;
            ray.z *= ray_length;

            RGBF color = trace_ray_iterative(device_scene.camera.pos, ray, &random);

            result.r += color.r;
            result.g += color.g;
            result.b += color.b;
        }

        float weight = 1.0f/(float)device_diffuse_samples;

        result.r *= weight;
        result.g *= weight;
        result.b *= weight;

        device_frame[id] = result;

        id += blockDim.x * gridDim.x;

        atomicAdd((uint32_t*)progress, 1);
        __threadfence_system();
    }
}

__global__
void set_up_raytracing_device() {
    device_amount = device_width * device_height;

    device_step = 2.0f * (device_scene.camera.fov / device_width);

    device_vfov = device_step * device_height / 2.0f;

    device_offset_x = (device_step / 2.0f);
    device_offset_y = (device_step / 2.0f);

    float alpha = device_scene.camera.rotation.x;
    float beta = device_scene.camera.rotation.y;
    float gamma = device_scene.camera.rotation.z;

    double cy = __cosf(gamma * 0.5);
    double sy = __sinf(gamma * 0.5);
    double cp = __cosf(beta * 0.5);
    double sp = __sinf(beta * 0.5);
    double cr = __cosf(alpha * 0.5);
    double sr = __sinf(alpha * 0.5);

    Quaternion q;
    q.w = cr * cp * cy + sr * sp * sy;
    q.x = sr * cp * cy - cr * sp * sy;
    q.y = cr * sp * cy + sr * cp * sy;
    q.z = cr * cp * sy - sr * sp * cy;

    device_camera_rotation = q;

    hiprand_init(0,0,0,&device_random);
}


extern "C" raytrace_instance* init_raytracing(const unsigned int width, const unsigned int height, const int reflection_depth, const int diffuse_samples) {
    raytrace_instance* instance = (raytrace_instance*)malloc(sizeof(raytrace_instance));

    instance->width = width;
    instance->height = height;
    instance->frame_buffer = (RGBF*)_mm_malloc(sizeof(RGBF) * width * height, 32);

    hipMalloc((void**) &(instance->frame_buffer_gpu), sizeof(RGBF) * width * height);

    instance->reflection_depth = reflection_depth;
    instance->diffuse_samples = diffuse_samples;

    return instance;
}

extern "C" void* initialize_textures(TextureRGBA* textures, const int textures_length) {
    hipTextureObject_t* textures_cpu = (hipTextureObject_t*) malloc(sizeof(hipTextureObject_t) * textures_length);
    hipTextureObject_t* textures_gpu;

    gpuErrchk(hipMalloc((void**) &(textures_gpu), sizeof(hipTextureObject_t) * textures_length));

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0]   = hipAddressModeWrap;
    texDesc.addressMode[1]   = hipAddressModeWrap;
    texDesc.filterMode       = hipFilterModeLinear;
    texDesc.readMode         = hipReadModeElementType;
    texDesc.normalizedCoords = 1;

    for (int i = 0; i < textures_length; i++) {
        TextureRGBA texture = textures[i];

        const int num_rows = texture.height;
        const int num_cols = texture.width;
        RGBAF* data = texture.data;
        RGBAF* data_gpu;
        size_t pitch;
        gpuErrchk(hipMallocPitch((void**) &data_gpu, &pitch, num_cols * sizeof(RGBAF), num_rows));
        gpuErrchk(hipMemcpy2D(data_gpu, pitch, data, num_cols * sizeof(RGBAF), num_cols * sizeof(RGBAF), num_rows, hipMemcpyHostToDevice));

        struct hipResourceDesc resDesc;
        memset(&resDesc, 0, sizeof(resDesc));
        resDesc.resType = hipResourceTypePitch2D;
        resDesc.res.pitch2D.devPtr = data_gpu;
        resDesc.res.pitch2D.width = num_cols;
        resDesc.res.pitch2D.height = num_rows;
        resDesc.res.pitch2D.desc = hipCreateChannelDesc<float4>();
        resDesc.res.pitch2D.pitchInBytes = pitch;

        gpuErrchk(hipCreateTextureObject(textures_cpu + i, &resDesc, &texDesc, NULL));
    }

    gpuErrchk(hipMemcpy(textures_gpu, textures_cpu, sizeof(hipTextureObject_t) * textures_length, hipMemcpyHostToDevice));

    free(textures_cpu);

    return textures_gpu;
}

extern "C" void initialize_device() {
    gpuErrchk(hipSetDeviceFlags(hipDeviceMapHost));
}

extern "C" void free_textures(void* texture_atlas, const int textures_length) {
    hipTextureObject_t* textures_cpu = (hipTextureObject_t*) malloc(sizeof(hipTextureObject_t) * textures_length);
    gpuErrchk(hipMemcpy(textures_cpu, texture_atlas, sizeof(hipTextureObject_t) * textures_length, hipMemcpyDeviceToHost));

    for (int i = 0; i < textures_length; i++) {
        gpuErrchk(hipDestroyTextureObject(textures_cpu[i]));
    }

    gpuErrchk(hipFree(texture_atlas));
    free(textures_cpu);
}

extern "C" void trace_scene(Scene scene, raytrace_instance* instance, void* albedo_atlas, void* illuminance_atlas, void* material_atlas, texture_assignment* texture_assignments, int meshes_count) {
    Scene scene_gpu = scene;

    volatile uint32_t *progress_gpu, *progress_cpu;

    hipEvent_t kernelFinished;
    gpuErrchk(hipEventCreate(&kernelFinished));
    gpuErrchk(hipHostAlloc((void**)&progress_cpu, sizeof(uint32_t), hipHostMallocMapped));
    gpuErrchk(hipHostGetDevicePointer((uint32_t**)&progress_gpu, (uint32_t*)progress_cpu, 0));
    *progress_cpu = 0;


    texture_assignment* texture_assignments_gpu;

    gpuErrchk(hipMalloc((void**) &(texture_assignments_gpu), sizeof(texture_assignment) * meshes_count));
    gpuErrchk(hipMalloc((void**) &(scene_gpu.triangles), sizeof(Triangle) * scene_gpu.triangles_length));
    gpuErrchk(hipMalloc((void**) &(scene_gpu.nodes), sizeof(Node) * scene_gpu.nodes_length));

    gpuErrchk(hipMemcpy(texture_assignments_gpu, texture_assignments, sizeof(texture_assignment) * meshes_count, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(scene_gpu.triangles, scene.triangles, sizeof(Triangle) * scene.triangles_length, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(scene_gpu.nodes, scene.nodes, sizeof(Node) * scene.nodes_length, hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_texture_assignments), &(texture_assignments_gpu), sizeof(texture_assignment*), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_frame), &(instance->frame_buffer_gpu), sizeof(RGBF*), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_scene), &scene_gpu, sizeof(Scene), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_reflection_depth), &(instance->reflection_depth), sizeof(int), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_diffuse_samples), &(instance->diffuse_samples), sizeof(int), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_width), &(instance->width), sizeof(unsigned int), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_height), &(instance->height), sizeof(unsigned int), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_albedo_atlas), &(albedo_atlas), sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_illuminance_atlas), &(illuminance_atlas), sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(device_material_atlas), &(material_atlas), sizeof(hipTextureObject_t), 0, hipMemcpyHostToDevice));


    set_up_raytracing_device<<<1,1>>>();

    gpuErrchk(hipDeviceSynchronize());

    clock_t t = clock();

    trace_rays<<<blocks_per_grid,threads_per_block>>>(progress_gpu);

    gpuErrchk(hipEventRecord(kernelFinished));

    uint32_t progress = 0;
    const uint32_t total_pixels = instance->width * instance->height;
    const float ratio = 1.0f/((float)instance->width * (float)instance->height);
    while (hipEventQuery(kernelFinished) != hipSuccess) {
        std::this_thread::sleep_for(std::chrono::microseconds(100000));
        uint32_t new_progress = *progress_cpu;
        if (new_progress > progress) {
            progress = new_progress;
            clock_t curr_time = clock();
            double time_elapsed = (((double)curr_time - t)/CLOCKS_PER_SEC);
            printf("\r                                                                                         \rProgress: %2.1f%% - Time Elapsed: %.1fs - Time Remaining: %.1fs",(float)progress * ratio * 100, time_elapsed, (total_pixels - progress) * (time_elapsed/progress));
        }
    }

    printf("\r                                                                             \r");

    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(instance->frame_buffer, instance->frame_buffer_gpu, sizeof(RGBF) * instance->width * instance->height, hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(texture_assignments_gpu));
    gpuErrchk(hipFree(scene_gpu.triangles));
    gpuErrchk(hipFree(scene_gpu.nodes));
}

extern "C" void free_raytracing(raytrace_instance* instance) {
    gpuErrchk(hipFree(instance->frame_buffer_gpu));

    _mm_free(instance->frame_buffer);

    free(instance);
}
