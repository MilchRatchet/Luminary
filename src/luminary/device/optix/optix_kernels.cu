#include "hip/hip_runtime.h"
#define OPTIX_KERNEL

#include "bvh_utils.cuh"
#include "math.cuh"
#include "memory.cuh"
#include "trace.cuh"
#include "utils.cuh"

enum OptixAlphaResult {
  OPTIX_ALPHA_RESULT_OPAQUE      = 0,
  OPTIX_ALPHA_RESULT_SEMI        = 1,
  OPTIX_ALPHA_RESULT_TRANSPARENT = 2
} typedef OptixAlphaResult;

// Kernels must be named __[SEMANTIC]__..., for example, __raygen__...
// This can be found under function name prefix in the programming guide

extern "C" __global__ void __raygen__optix() {
  const uint16_t trace_task_count = device.ptrs.trace_counts[THREAD_ID];

  for (int i = 0; i < trace_task_count; i++) {
    const int offset         = get_task_address(i);
    const TraceTask task     = load_trace_task(device.ptrs.trace_tasks + offset);
    const TraceResult result = trace_preprocess(task);

    const float3 origin = make_float3(task.origin.x, task.origin.y, task.origin.z);
    const float3 ray    = make_float3(task.ray.x, task.ray.y, task.ray.z);

    const float tmax = result.depth;

    unsigned int depth  = __float_as_uint(result.depth);
    unsigned int hit_id = result.hit_id;

    optixTrace(device.optix_bvh, origin, ray, 0.0f, tmax, 0.0f, OptixVisibilityMask(0xFFFF), 0, 0, 0, 0, depth, hit_id);

    float2 trace_result;

    if (device.settings.shading_mode == LUMINARY_SHADING_MODE_HEAT) {
      trace_result = make_float2(0.0f, __uint_as_float(hit_id));
    }
    else {
      trace_result = make_float2(__uint_as_float(depth), __uint_as_float(hit_id));
    }

    __stcs((float2*) (device.ptrs.trace_results + offset), trace_result);
  }
}

// TODO: This is fucked, I need to fix it by checking for IOR aswell.
#if 0
/*
 * Performs alpha test on triangle
 * @result 0 if opaque, 1 if transparent, 2 if alpha cutoff
 */
__device__ OptixAlphaResult optix_alpha_test() {
  const unsigned int hit_id = optixGetPrimitiveIndex();

  const uint32_t material_id = load_triangle_material_id(hit_id);
  const uint16_t tex         = __ldg(&(device.ptrs.materials[material_id].albedo_tex));

  if (tex != TEXTURE_NONE) {
    const UV uv = load_triangle_tex_coords(hit_id, optixGetTriangleBarycentrics());

    const float alpha = tex2D<float4>(device.ptrs.albedo_atlas[tex].handle, uv.u, 1.0f - uv.v).w;

    if (alpha == 0.0f) {
      return OPTIX_ALPHA_RESULT_TRANSPARENT;
    }

    if (alpha < 1.0f) {
      return OPTIX_ALPHA_RESULT_SEMI;
    }
  }

  return OPTIX_ALPHA_RESULT_OPAQUE;
}

extern "C" __global__ void __anyhit__optix() {

  const OptixAlphaResult alpha_result = optix_alpha_test();

  if (alpha_result == OPTIX_ALPHA_RESULT_TRANSPARENT) {
    optixIgnoreIntersection();
  }
}
#else
extern "C" __global__ void __anyhit__optix() {
  return;
}
#endif

extern "C" __global__ void __closesthit__optix() {
  optixSetPayload_0(__float_as_uint(optixGetRayTmax()));
  optixSetPayload_1(optixGetPrimitiveIndex());
}
